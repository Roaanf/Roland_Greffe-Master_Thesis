#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// Gigavoxels
#include <GvCore/GsVector.h>
#include <GvStructure/GsVolumeTree.h>
#include <GvStructure/GsDataProductionManager.h>
#include <GvRendering/GsRendererCUDA.h>
#include <GvUtils/GsSimplePipeline.h>
#include <GvUtils/GsSimpleHostShader.h>
#include <GvUtils/GsSimplePriorityPoliciesManagerKernel.h>
#include <GvUtils/GsDataLoader.h>
#include <GvUtils/GsCommonGraphicsPass.h>
#include <GvCore/GsError.h>
#include <GvPerfMon/GsPerformanceMonitor.h>
#include <GvStructure/GsNode.h>
#include <GvUtils/GsEnvironment.h>

// Project
#include "Producer.h"
#include "ShaderKernel.h"

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 1024U * 1024U * 1024U )		// 1024 Mo
#define BRICKPOOL_MEMSIZE	( 2048U * 1024U * 1024U )		// 2048 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	_renderer( NULL )
,	mColorTex( 0 )
,	mDepthTex( 0 )
,	_frameBuffer( 0 )
,	_inputDepthBuffer( 0 )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 16 )
{
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
	//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
	hipSetDevice( gpuGetMaxGflopsDeviceId() );
	GV_CHECK_CUDA_ERROR( "hipSetDevice" );

	// Compute the size of one element in the cache for nodes and bricks
	size_t nodeElemSize = PipelineType::NodeTileResolution::numElements * sizeof( GvStructure::GsNode );
	//size_t brickElemSize = PipelineType::RealBrickTileResolution::numElements * GvCore::DataTotalChannelSize< PipelineType::DataTypeList >::value;

	// Compute how many we can fit into the given memory size
	size_t nodePoolNumElems = NODEPOOL_MEMSIZE / nodeElemSize;
	//size_t brickPoolNumElems = BRICKPOOL_MEMSIZE / brickElemSize;

	// Compute the resolution of the pools
	uint3 nodePoolRes = make_uint3((uint)floorf(powf((float)nodePoolNumElems, 1.0f / 3.0f))) * NodeRes::get();
	//uint3 brickPoolRes = make_uint3((uint)floorf(powf((float)brickPoolNumElems, 1.0f / 3.0f))) * RealBrickRes::get();

	//std::cout << "" << std::endl;
	//std::cout << "nodePoolRes: " << nodePoolRes << std::endl;
	//std::cout << "brickPoolRes: " << brickPoolRes << std::endl;

	// Pipeline creation
	_pipeline = new PipelineType();

	// Producer creation
	//QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
	//QString filename = dataRepository + QDir::separator() + QString( "Voxels" ) + QDir::separator() + QString( "xyzrgb_dragon512_BR8_B1" ) + QDir::separator() + QString( "xyzrgb_dragon.xml" );
	QString filename = GsEnvironment::getDataDir( GsEnvironment::eVoxelsDir ).c_str();
	filename += QDir::separator();
	filename += QString( "xyzrgb_dragon512_BR8_B1" );
	filename += QDir::separator();
	filename += QString( "xyzrgb_dragon.xml" );
	GvUtils::GsDataLoader< DataType >* dataLoader = new GvUtils::GsDataLoader< DataType >(
														filename.toStdString(),
														PipelineType::BrickTileResolution::get(), PipelineType::BrickTileBorderSize, true );
													//	make_uint3( 512 ), BrickRes::get(), BrickBorderSize, false );
		
	// Shader creation
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, shader );

	// Producer initialization
	_producer = new ProducerType( 64 * 1024 * 1024, nodePoolRes.x * nodePoolRes.y * nodePoolRes.z );
	assert( _producer != NULL );
	_producer->attachProducer( dataLoader );
	_pipeline->addProducer( _producer );

	// Renderer initialization
	_renderer = new RendererType( _pipeline->editDataStructure(), _pipeline->editCache() );
	assert( _renderer != NULL );
	_pipeline->addRenderer( _renderer );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
	_pipeline->editCache()->setMaxNbNodeSubdivisions( 500 );
	_pipeline->editCache()->setMaxNbBrickLoads( 300 );
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( PipelineType::CacheType::NodesCacheManager::eAllPolicies );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( PipelineType::CacheType::BricksCacheManager::eAllPolicies );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );

	CUDAPM_START_EVENT( app_init_frame );

	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );

	glMatrixMode( GL_MODELVIEW );

	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		glTranslatef( -0.5f, -0.5f, -0.5f );
		_pipeline->editDataStructure()->render();
		glPopMatrix();
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _inputDepthBuffer );
		glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		glClear( GL_COLOR_BUFFER_BIT );
	}

	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv(GL_MODELVIEW_MATRIX, viewMatrix._array);
	glGetFloatv(GL_PROJECTION_MATRIX, projectionMatrix._array);

	// build and extract tree transformations
	float4x4 modelMatrix;

	glPushMatrix();
	glLoadIdentity();
	glTranslatef(-0.5f, -0.5f, -0.5f);
	glGetFloatv(GL_MODELVIEW_MATRIX, modelMatrix._array);
	glPopMatrix();

	// extract viewport
	GLint params[4];
	glGetIntegerv(GL_VIEWPORT, params);
	int4 viewport = make_int4(params[0], params[1], params[2], params[3]);

	CUDAPM_STOP_EVENT( app_init_frame );

	// render the scene into textures
	_pipeline->execute(modelMatrix, viewMatrix, projectionMatrix, viewport);

	// Render the result to the screen
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();

	glEnable(GL_TEXTURE_RECTANGLE_EXT);
	glDisable(GL_DEPTH_TEST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, mColorTex );

	GLint sMin = 0;
	GLint tMin = 0;
	GLint sMax = _width;
	GLint tMax = _height;

	glBegin(GL_QUADS);
		glColor3f(1.0f, 1.0f, 1.0f);
		glTexCoord2i(sMin, tMin); glVertex2i(-1, -1);
		glTexCoord2i(sMax, tMin); glVertex2i( 1, -1);
		glTexCoord2i(sMax, tMax); glVertex2i( 1,  1);
		glTexCoord2i(sMin, tMax); glVertex2i(-1,  1);
	glEnd();

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);

	glDisable(GL_TEXTURE_RECTANGLE_EXT);

	glPopMatrix();
	glMatrixMode(GL_MODELVIEW);
	glPopMatrix();

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	///*_pipeline->editRenderer()*/_renderer->doPostRender();
		
	//mVolumeTreeRenderer->nextFrame();
	/*_pipeline->editRenderer()*/_renderer->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	if (_displayPerfmon)
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL(_displayPerfmon - 1);
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	_width = width;
	_height = height;

	// Reset default active frame region for rendering
	/*_pipeline->editRenderer()*/_renderer->setProjectedBBox( make_uint4( 0, 0, _width, _height ) );
	
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE(make_uint2(_width, _height));

	/*uchar *timersMask = GvPerfMon::CUDAPerfMon::get().getKernelTimerMask();
	hipMemset(timersMask, 255, _width * _height);*/

	// Create frame-dependent objects

	// Disconnect all registered graphics resources
	/*_pipeline->editRenderer()*/_renderer->resetGraphicsResources();
	
	// ...
	if (_inputDepthBuffer)
		glDeleteBuffers(1, &_inputDepthBuffer);

	if (mColorTex)
		glDeleteTextures(1, &mColorTex);
	if (mDepthTex)
		glDeleteTextures(1, &mDepthTex);

	if (_frameBuffer)
		glDeleteFramebuffers(1, &_frameBuffer);

	glGenTextures(1, &mColorTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mColorTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenBuffers(1, &_inputDepthBuffer);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, _inputDepthBuffer);
	glBufferData(GL_PIXEL_PACK_BUFFER, width * height * sizeof(GLuint), NULL, GL_DYNAMIC_DRAW);	// modifier le param�tre du depth PBO pour de la elcture seule !!! STREAM_READ
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glGenTextures(1, &mDepthTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mDepthTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_DEPTH24_STENCIL8_EXT, width, height, 0, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenFramebuffers( 1, &_frameBuffer );
	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_RECTANGLE_EXT, mColorTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, mDepthTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_STENCIL_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, mDepthTex, 0 );
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );
	GV_CHECK_GL_ERROR();

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, mColorTex, GL_TEXTURE_RECTANGLE_EXT );
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _inputDepthBuffer );
	}
	else
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, mColorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	/*_pipeline->editRenderer()*/_renderer->resetGraphicsResources();

	if ( _displayOctree )
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, mColorTex, GL_TEXTURE_RECTANGLE_EXT );
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _inputDepthBuffer );
	}
	else
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, mColorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * ...
 *
 * @param mode ...
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if (_displayPerfmon)
		_displayPerfmon = 0;
	else
		_displayPerfmon = mode;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if (_maxVolTreeDepth < 32)
		_maxVolTreeDepth++;

	//mVolumeTree->setMaxDepth( _maxVolTreeDepth );
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if (_maxVolTreeDepth > 0)
		_maxVolTreeDepth--;

	//mVolumeTree->setMaxDepth( _maxVolTreeDepth );
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	/*_pipeline->editRenderer()*/_renderer->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	// WARNING
	// Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
	// Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
	float3 translation = make_float3( -0.5f, -0.5f, -0.5f );
	float3 lightPosition = make_float3( pX, pY, pZ ) - translation;
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &lightPosition, sizeof( lightPosition ), 0, hipMemcpyHostToDevice ) );
}
