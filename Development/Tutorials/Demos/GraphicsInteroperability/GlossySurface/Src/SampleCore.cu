#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvStructure/GvDataProductionManager.h>
#include <GvUtils/GvSimplePipeline.h>
#include <GvUtils/GvSimpleHostProducer.h>
#include <GvUtils/GvDataLoader.h>
#include <GvUtils/GvSimpleHostShader.h>
#include <GvUtils/GvSimplePriorityPoliciesManagerKernel.h>
#include <GvUtils/GvCommonGraphicsPass.h>
#include <GvCore/GvError.h>
#include <GvPerfMon/GvPerformanceMonitor.h>
#include <GvStructure/GvNode.h>

// Project
#include "Producer.h"
//#include "VolumeProducerBricks.h"
#include "VolumeTreeRendererGLSL.h"
//#include "ProducerTorusKernel.h"

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>



/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/
GlossyObject* glossyObject = NULL;
Mesh* environmentObject = NULL;
CubeMap* cubeMap = NULL;
float scale = 1.0;
float translation[3] = {0.0};

/**
 * Defines the size allowed for each type of pool
 */
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 128U * 1024U * 1024U )	// 128 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	mDisplayOctree( false )
,	mDisplayPerfmon( 0 )
,	mMaxVolTreeDepth( 16 )
{
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init(SampleViewer* sv)
{
	CUDAPM_INIT();
	hipSetDevice( gpuGetMaxGflopsDeviceId() );
	GV_CHECK_CUDA_ERROR( "hipSetDevice" );

	// Compute the size of one element in the cache for nodes and bricks
	size_t nodeElemSize = NodeRes::numElements * sizeof( GvStructure::GvNode );
	size_t brickElemSize = RealBrickRes::numElements * GvCore::DataTotalChannelSize< DataType >::value;

	// Compute how many we can fit into the given memory size
	size_t nodePoolNumElems = NODEPOOL_MEMSIZE / nodeElemSize;
	size_t brickPoolNumElems = BRICKPOOL_MEMSIZE / brickElemSize;

	// Compute the resolution of the pools
	uint3 nodePoolRes = make_uint3( (uint)floorf( powf( (float)nodePoolNumElems, 1.0f / 3.0f ) ) ) * NodeRes::get();

	// Pipeline creation
	_pipeline = new PipelineType();

	// Producer creation
	ProducerType* producer = new ProducerType( 64 * 1024 * 1024, nodePoolRes.x * nodePoolRes.y * nodePoolRes.z );
	
	// Shader creation
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	const bool useGraphicsLibraryInteroperability = true;
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, producer, shader, useGraphicsLibraryInteroperability );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );

	//getting the sample viewer
	sviewer = sv;

	/****************CREATING GLOSSY OBJECT****************/
	glossyObject = new GlossyObject();
	_pipeline->editRenderer()->getVolTreeChildArray()->unmapResource();
	_pipeline->editRenderer()->getVolTreeDataArray()->unmapResource();
	glossyObject->setVolTreeChildArray(_pipeline->editRenderer()->getVolTreeChildArray(), _pipeline->editRenderer()->getChildBufferName());
	glossyObject->setVolTreeDataArray(_pipeline->editRenderer()->getVolTreeDataArray(), _pipeline->editRenderer()->getDataBufferName());	
	glossyObject->init();
	_pipeline->editRenderer()->getVolTreeChildArray()->mapResource();
	_pipeline->editRenderer()->getVolTreeDataArray()->mapResource();
	
	/****************CREATING CUBE MAP****************/
	/*cubeMap = new CubeMap( "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posx.jpg",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negx.jpg",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posy.jpg",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negy.jpg",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posz.jpg",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negz.jpg");*/
	cubeMap = new CubeMap( "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posx.png",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negx.png",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posy.png",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negy.png",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/posz.png",
						   "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/CubeMapTextures/NissiBeach2/negz.png");
	cubeMap->Load();
	cubeMap->init();

	/****************CREATING ENVIRONMENT******************/
	GLuint vshader = useShader(GL_VERTEX_SHADER, "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/Res/vert.glsl");
	GLuint fshader = useShader(GL_FRAGMENT_SHADER, "../../Development/Tutorials/Demos/GraphicsInteroperability/GlossySurface/Res/frag.glsl");
	GLuint program = glCreateProgram();
	glAttachShader(program, vshader);
	glAttachShader(program, fshader);
	glLinkProgram(program);
	linkStatus(program);
	environmentObject = new Mesh(program);
	//string environmentObjectFile = "Data/3DModels/MickeyMouse.obj";	
	string environmentObjectFile = "Data/3DModels/dino.3ds";	
	environmentObject->chargerMesh(environmentObjectFile);
	environmentObject->creerVBO();
	scale = environmentObject->getScaleFactor();
	environmentObject->getTranslationFactors(translation);

	/************************VOXELIZATION***********************/
	//voxelization code on file environmentObjectFile
	//...
	//...
	/*************************************************************/

	//retrieving the xml file 
	QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
	QString filename = dataRepository + QDir::separator() + QString( "Voxels" ) + QDir::separator() + QString( /*"xyzrgb_dragon512_BR8_B1"*/"Dino") + QDir::separator() + QString( /*"xyzrgb_dragon.xml"*/"dino.xml");
	GvUtils::GvDataLoader< DataType >* dataLoader = new GvUtils::GvDataLoader< DataType >( filename.toStdString(), BrickRes::get(), BrickBorderSize, true );
	producer->attachProducer( dataLoader );

	glEnable(GL_DEPTH_TEST);
	//glEnable(GL_CULL_FACE);
	//glCullFace(GL_FRONT);
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	glClearColor( 0.0f, 0.1f, 0.3f, 0.0f );
	glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );


	glMatrixMode( GL_MODELVIEW);

	// Display the data structure (space partitioning)
	if ( mDisplayOctree )
	{
		glPushMatrix();
		glTranslatef(translation[0], translation[1], translation[2]);
		glScalef(scale, scale, scale);
		glTranslatef( -0.5f, -0.5f, -0.5f );//pour centrer la boite GV
		_pipeline->editDataStructure()->render();
		glPopMatrix();
	}

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	// FIXME
	glPushMatrix();
	glTranslatef(translation[0], translation[1], translation[2]);
	glScalef(scale, scale, scale);
	glTranslatef( -0.5f, -0.5f, -0.5f );//pour centrer la boite GV
	// FIXME
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );
	// FIXME
	glPopMatrix();

	// build and extract tree transformations
	float4x4 modelMatrix;

	glPushMatrix();
	glLoadIdentity();
	glTranslatef(translation[0], translation[1], translation[2]);
	glScalef(scale, scale, scale);
	glTranslatef( -0.5f, -0.5f, -0.5f);//pour centrer la boite GV
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// extract viewport
	GLint params[4];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4( params[0], params[1], params[2], params[3] );

	CUDAPM_STOP_EVENT( app_init_frame );
	
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	/******************RENDER GIGAVOXELS SCENE******************/
	_pipeline->editRenderer()->setLightPosition(lightPos.x, lightPos.y, lightPos.z);
	_pipeline->execute( modelMatrix, viewMatrix, projectionMatrix, viewport );

	/******************RENDER THE CUBE MAP*********************/
	cubeMap->render();
	float cubeModelMatrix[16];
	cubeMap->getCubeModelMatrix(cubeModelMatrix);

	glEnable(GL_DEPTH_TEST);
	
	/******************RENDER THE GLOSSY OBJECT AND RETRIEVE THE GIGAVOXELS INFO*********************/
	glossyObject->setModelMatrix(modelMatrix._array[0],modelMatrix._array[1], modelMatrix._array[2], modelMatrix._array[3],  
							modelMatrix._array[4],modelMatrix._array[5], modelMatrix._array[6], modelMatrix._array[7], 
							modelMatrix._array[8],modelMatrix._array[9], modelMatrix._array[10], modelMatrix._array[11],
							modelMatrix._array[12],modelMatrix._array[13], modelMatrix._array[14], modelMatrix._array[15]);

	glossyObject->setCubeModelMatrix(cubeModelMatrix);
	glossyObject->setLightPosition(lightPos.x, lightPos.y, lightPos.z);
	glossyObject->setWorldLight(worldLight.x, worldLight.y, worldLight.z);
	glossyObject->setWorldCameraPosition(worldCamPos.x, worldCamPos.y, worldCamPos.z);
	glossyObject->setTexBufferName(_pipeline->editRenderer()->getTexBufferName());
	glossyObject->setCubeMapTextureID(cubeMap->getTextureID());
	uint3 bsc = _pipeline->editRenderer()->getBrickCacheSize();
	glossyObject->setBrickCacheSize(bsc.x, bsc.y, bsc.z);
	float3 bpri = _pipeline->editRenderer()->getBrickPoolResInv();
	glossyObject->setBrickPoolResInv(bpri.x, bpri.y, bpri.z);
	glossyObject->setMaxDepth(_pipeline->editRenderer()->getMaxDepth());
	GV_CHECK_GL_ERROR();
	glossyObject->render();
	GV_CHECK_GL_ERROR();
	
	/******************RENDER THE OBJECT CASTING ITS REFLECTION*******************/
	environmentObject->setLightPosition(lightPos.x, lightPos.y, lightPos.z);
	environmentObject->render();



	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	_pipeline->editRenderer()->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the performance monitor
	if ( mDisplayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL( mDisplayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	mWidth = width;
	mHeight = height;

	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( mWidth, mHeight ) );

	/*uchar *timersMask = GvPerfMon::CUDAPerfMon::get().getKernelTimerMask();
	hipMemset(timersMask, 255, mWidth * mHeight);*/
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	mDisplayOctree = !mDisplayOctree;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * ...
 *
 * @param mode ...
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( mDisplayPerfmon )
	{
		mDisplayPerfmon = 0;
	}
	else
	{
		mDisplayPerfmon = mode;
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( mMaxVolTreeDepth < 32 )
	{
		mMaxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( mMaxVolTreeDepth > 0 )
	{
		mMaxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * Set the light position in camera coordinates
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{	
	lightPos = make_float3( pX, pY, pZ );
}

/******************************************************************************
 * Set the light position in world coordinates
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setWorldLight( float pX, float pY, float pZ ) {
	worldLight = make_float3(pX, pY, pZ);
}

void SampleCore::setWorldCamera(float x, float y, float z) {
	worldCamPos.x = x;
	worldCamPos.y = y;
	worldCamPos.z = z;
} 

/******************************************************************************
 * Returns the file name of the shadow casting object (OpenGL)
 ******************************************************************************/
string SampleCore::getShadowCasterFile() {
	return shadowCasterFile;
}
