#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/GsVector.h>
#include <GvStructure/GsVolumeTree.h>
#include <GvStructure/GsDataProductionManager.h>
#include <GvUtils/GsSimplePipeline.h>
#include <GvUtils/GsSimpleHostProducer.h>
#include <GvUtils/GsSimpleHostShader.h>
#include <GvUtils/GsSimplePriorityPoliciesManagerKernel.h>
#include <GvUtils/GsCommonGraphicsPass.h>
#include <GvCore/GsError.h>
#include <GvPerfMon/GsPerformanceMonitor.h>

// Project
#include "ProducerKernel.h"
#include "ShaderKernel.h"
#include "RendererCUDA.h"
#include "ProxyGeometry.h"
#include "Mesh.h"

// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;
using namespace GsGraphics;

// GigaVoxels viewer
using namespace GvViewerCore;

// STL
using namespace std;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	_producer( NULL )
,	_renderer( NULL )
,	_graphicsEnvironment( NULL )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 5 )
,	_depthBuffer( 0 )
,	_colorTex( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_width( 0 )
,	_height( 0 )
,	_proxyGeometry( NULL )
{
	// Translation used to position the GigaVoxels data structure
	_translation[ 0 ] = -0.5f;
	_translation[ 1 ] = -0.5f;
	_translation[ 2 ] = -0.5f;

	// Light position
	_lightPosition = make_float3( 1.f, 1.f, 1.f );
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
	delete _graphicsEnvironment;

	delete _proxyGeometry;
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "Proxy Geometry Manager";
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Pipeline creation
	_pipeline = new PipelineType();
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, shader );

	// Producer initialization
	_producer = new ProducerType();
	assert( _producer != NULL );
	_pipeline->addProducer( _producer );

	// Renderer initialization
	_renderer = new RendererType( _pipeline->editDataStructure(), _pipeline->editCache() );
	assert( _renderer != NULL );
	_pipeline->addRenderer( _renderer );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );

	// Configure the Cache Management System
	_pipeline->editCache()->setMaxNbNodeSubdivisions( 500 );
	_pipeline->editCache()->setMaxNbBrickLoads( 300 );
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( DataProductionManagerType::NodesCacheManager::eAllPolicies );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( DataProductionManagerType::BricksCacheManager::eAllPolicies );
	
	// Graphics environment creation
	_graphicsEnvironment = new GsCommonGraphicsPass();

	// Initialize proxy geometry
	//
	// - find a way to modify internal buffer size
	_proxyGeometry = new ProxyGeometry();
	const QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
	const QString meshRepository = dataRepository + QDir::separator() + QString( "3DModels" );
	const QString meshFilename = meshRepository + QDir::separator() + QString( "bunny.obj" );
	_proxyGeometry->set3DModelFilename( meshFilename.toStdString() );
	_proxyGeometry->initialize();
	// Restore previous proxy geometry state
	_proxyGeometry->setScreenBasedCriteria( false );
	_proxyGeometry->setScreenBasedCriteriaCoefficient( 1.f );
	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( 512.f );
	// Register proxy geometry
	/*_pipeline->editRenderer()*/_renderer->setProxyGeometry( _proxyGeometry );

	// Noise parameters
	setNoiseFirstFrequency( 3.61f );
	setNoiseStrength( 44.8f );
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	glMatrixMode( GL_MODELVIEW );
	
	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		glTranslatef( -0.5f, -0.5f, -0.5f );
		_pipeline->editDataStructure()->render();
		glPopMatrix();
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
		glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		//glClear( GL_COLOR_BUFFER_BIT );
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );
	}
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

	// extract viewport
	GLint params[ 4 ];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4( params[ 0 ], params[ 1 ], params[ 2 ], params[ 3 ] );

	// render the scene into textures
	CUDAPM_STOP_EVENT( app_init_frame );

	// Generate depth maps from mesh
	// - min depth from closest faces
	// - max depth from farthest faces
	// => we get a "shell" from the mesh
	glEnable( GL_DEPTH_TEST );
	glDisable( GL_CULL_FACE );
	float4x4 proxyGeometryModelViewMatrix;
	glPushMatrix();
	// Add Model transformation to lie between -0.5 and 0.5
	const IMesh* mesh = _proxyGeometry->getMesh();
	const float minX = mesh->_minX;
	const float minY = mesh->_minY;
	const float minZ = mesh->_minZ;
	const float maxX = mesh->_maxX;
	const float maxY = mesh->_maxY;
	const float maxZ = mesh->_maxZ;
	const float uniformScale = 0.99f / std::max( std::max( maxX - minX, maxY - minY ), maxZ - minZ );
	glScalef( uniformScale, uniformScale, uniformScale );
	const float3 translate = make_float3( - ( minX + maxX ) * 0.5f, - ( minY + maxY ) * 0.5f, - ( minZ + maxZ ) * 0.5f );
	glTranslatef( translate.x, translate.y, translate.z );
	glGetFloatv( GL_MODELVIEW_MATRIX, proxyGeometryModelViewMatrix._array );
	// TO DO : add a screen based criteria to stop division => ...
	_proxyGeometry->render( proxyGeometryModelViewMatrix, projectionMatrix, viewport );
	glPopMatrix();

	// Build the world transformation matrix
	float4x4 modelMatrix;
	glPushMatrix();
	glLoadIdentity();
	glTranslatef( -0.5f, -0.5f, -0.5f );
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// Render
	_pipeline->execute( modelMatrix, viewMatrix, projectionMatrix, viewport );

	// Render the result to the screen
	glMatrixMode( GL_MODELVIEW );
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode( GL_PROJECTION );
	glPushMatrix();
	glLoadIdentity();

	glDisable( GL_DEPTH_TEST );
	glEnable( GL_TEXTURE_RECTANGLE_EXT );
	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );
	
	// Draw a full screen quad
	GLint sMin = 0;
	GLint tMin = 0;
	GLint sMax = _width;
	GLint tMax = _height;
	glBegin( GL_QUADS );
	glColor3f( 1.0f, 1.0f, 1.0f );
	glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
	glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
	glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
	glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
	glEnd();

	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );
	glDisable( GL_TEXTURE_RECTANGLE_EXT );
	
	glPopMatrix();
	glMatrixMode( GL_MODELVIEW );
	glPopMatrix();

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	///*_pipeline->editRenderer()*/_renderer->doPostRender();
	
	// Update GigaVoxels info
	/*_pipeline->editRenderer()*/_renderer->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the GigaVoxels performance monitor (if it has been activated during GigaVoxels compilation)
	if ( _displayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL( _displayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int pWidth, int pHeight )
{
	// LOG
	//
	// @todo : check and avoid 0 values, replace by 1 and warn user
	if ( pWidth == 0 )
	{
		// TO DO
		// ...
	}
	if ( pHeight == 0 )
	{
		// TO DO
		// ...
	}

	_width = pWidth;
	_height = pHeight;

	// Reset default active frame region for rendering
	/*_pipeline->editRenderer()*/_renderer->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );

	// Update graphics environment
	_graphicsEnvironment->setBufferSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Reset graphics resources
 ******************************************************************************/
void SampleCore::resetGraphicsresources()
{
	// [ 1 ] - Reset graphics resources

	// Disconnect all registered graphics resources
	/*_pipeline->editRenderer()*/_renderer->resetGraphicsResources();
	
	// Update graphics environment
	_graphicsEnvironment->reset();
	
	// Update internal variables
	_depthBuffer = _graphicsEnvironment->getDepthBuffer();
	_colorTex = _graphicsEnvironment->getColorTexture();
	//_colorRenderBuffer = _graphicsEnvironment->getColorRenderBuffer();
	_depthTex = _graphicsEnvironment->getDepthTexture();
	_frameBuffer = _graphicsEnvironment->getFrameBuffer();
	
	// [ 2 ] - Connect graphics resources

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}

	// Reset proxy geometry resources
	/*_pipeline->editRenderer()*/_renderer->unregisterProxyGeometryGraphicsResources();
	_proxyGeometry->setBufferSize( _width, _height );
	/*_pipeline->editRenderer()*/_renderer->registerProxyGeometryGraphicsResources();
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	/*_pipeline->editRenderer()*/_renderer->resetGraphicsResources();

	if ( _displayOctree )
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		/*_pipeline->editRenderer()*/_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Get the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::getDataStructureAppearance( bool& pShowNodeHasBrickTerminal, bool& pShowNodeHasBrickNotTerminal, bool& pShowNodeIsBrickNotInCache, bool& pShowNodeEmptyOrConstant
											, float& pNodeHasBrickTerminalColorR, float& pNodeHasBrickTerminalColorG, float& pNodeHasBrickTerminalColorB, float& pNodeHasBrickTerminalColorA
											, float& pNodeHasBrickNotTerminalColorR, float& pNodeHasBrickNotTerminalColorG, float& pNodeHasBrickNotTerminalColorB, float& pNodeHasBrickNotTerminalColorA
											, float& pNodeIsBrickNotInCacheColorR, float& pNodeIsBrickNotInCacheColorG, float& pNodeIsBrickNotInCacheColorB, float& pNodeIsBrickNotInCacheColorA
											, float& pNodeEmptyOrConstantColorR, float& pNodeEmptyOrConstantColorG, float& pNodeEmptyOrConstantColorB, float& pNodeEmptyOrConstantColorA ) const
{
	float4 nodeHasBrickTerminalColor;
	float4 nodeHasBrickNotTerminalColor;
	float4 nodeIsBrickNotInCacheColor;
	float4 nodeEmptyOrConstantColor;
										
	_pipeline->getDataStructure()->getDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );

	pNodeHasBrickTerminalColorR = nodeHasBrickTerminalColor.x;
	pNodeHasBrickTerminalColorG = nodeHasBrickTerminalColor.y;
	pNodeHasBrickTerminalColorB = nodeHasBrickTerminalColor.z;
	pNodeHasBrickTerminalColorA = nodeHasBrickTerminalColor.w;

	pNodeHasBrickNotTerminalColorR = nodeHasBrickNotTerminalColor.x;
	pNodeHasBrickNotTerminalColorG = nodeHasBrickNotTerminalColor.y;
	pNodeHasBrickNotTerminalColorB = nodeHasBrickNotTerminalColor.z;
	pNodeHasBrickNotTerminalColorA = nodeHasBrickNotTerminalColor.w;

	pNodeIsBrickNotInCacheColorR = nodeIsBrickNotInCacheColor.x;
	pNodeIsBrickNotInCacheColorG = nodeIsBrickNotInCacheColor.y;
	pNodeIsBrickNotInCacheColorB = nodeIsBrickNotInCacheColor.z;
	pNodeIsBrickNotInCacheColorA = nodeIsBrickNotInCacheColor.w;

	pNodeEmptyOrConstantColorR = nodeEmptyOrConstantColor.x;
	pNodeEmptyOrConstantColorG = nodeEmptyOrConstantColor.y;
	pNodeEmptyOrConstantColorB = nodeEmptyOrConstantColor.z;
	pNodeEmptyOrConstantColorA = nodeEmptyOrConstantColor.w;
}

/******************************************************************************
 * Set the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::setDataStructureAppearance( bool pShowNodeHasBrickTerminal, bool pShowNodeHasBrickNotTerminal, bool pShowNodeIsBrickNotInCache, bool pShowNodeEmptyOrConstant
											, float pNodeHasBrickTerminalColorR, float pNodeHasBrickTerminalColorG, float pNodeHasBrickTerminalColorB, float pNodeHasBrickTerminalColorA
											, float pNodeHasBrickNotTerminalColorR, float pNodeHasBrickNotTerminalColorG, float pNodeHasBrickNotTerminalColorB, float pNodeHasBrickNotTerminalColorA
											, float pNodeIsBrickNotInCacheColorR, float pNodeIsBrickNotInCacheColorG, float pNodeIsBrickNotInCacheColorB, float pNodeIsBrickNotInCacheColorA
											, float pNodeEmptyOrConstantColorR, float pNodeEmptyOrConstantColorG, float pNodeEmptyOrConstantColorB, float pNodeEmptyOrConstantColorA )
{
	float4 nodeHasBrickTerminalColor = make_float4( pNodeHasBrickTerminalColorR, pNodeHasBrickTerminalColorG, pNodeHasBrickTerminalColorB, pNodeHasBrickTerminalColorA );
	float4 nodeHasBrickNotTerminalColor = make_float4( pNodeHasBrickNotTerminalColorR, pNodeHasBrickNotTerminalColorG, pNodeHasBrickNotTerminalColorB, pNodeHasBrickNotTerminalColorA );
	float4 nodeIsBrickNotInCacheColor = make_float4( pNodeIsBrickNotInCacheColorR, pNodeIsBrickNotInCacheColorG, pNodeIsBrickNotInCacheColorB, pNodeIsBrickNotInCacheColorA );
	float4 nodeEmptyOrConstantColor = make_float4( pNodeEmptyOrConstantColorR, pNodeEmptyOrConstantColorG, pNodeEmptyOrConstantColorB, pNodeEmptyOrConstantColorA );

	_pipeline->editDataStructure()->setDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Get the node tile resolution of the data structure.
 *
 * @param pX the X node tile resolution
 * @param pY the Y node tile resolution
 * @param pZ the Z node tile resolution
 ******************************************************************************/
void SampleCore::getDataStructureNodeTileResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& nodeTileResolution = _pipeline->editDataStructure()->getNodeTileResolution().get();

	pX = nodeTileResolution.x;
	pY = nodeTileResolution.y;
	pZ = nodeTileResolution.z;
}

/******************************************************************************
 * Get the brick resolution of the data structure (voxels).
 *
 * @param pX the X brick resolution
 * @param pY the Y brick resolution
 * @param pZ the Z brick resolution
 ******************************************************************************/
void SampleCore::getDataStructureBrickResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& brickResolution = _pipeline->editDataStructure()->getBrickResolution().get();

	pX = brickResolution.x;
	pY = brickResolution.y;
	pZ = brickResolution.z;
}

/******************************************************************************
 * Get the max depth.
 *
 * @return the max depth
 ******************************************************************************/
unsigned int SampleCore::getRendererMaxDepth() const
{
	return _pipeline->editDataStructure()->getMaxDepth();
}

/******************************************************************************
 * Set the max depth.
 *
 * @param pValue the max depth
 ******************************************************************************/
void SampleCore::setRendererMaxDepth( unsigned int pValue )
{
	_pipeline->editDataStructure()->setMaxDepth( pValue );
}

/******************************************************************************
 * Get the max number of requests of node subdivisions.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbNodeSubdivisions() const
{
	return _pipeline->getCache()->getMaxNbNodeSubdivisions();
}

/******************************************************************************
 * Set the max number of requests of node subdivisions.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbNodeSubdivisions( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbNodeSubdivisions( pValue );
}

/******************************************************************************
 * Get the max number of requests of brick of voxel loads.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbBrickLoads() const
{
	return _pipeline->getCache()->getMaxNbBrickLoads();
}

/******************************************************************************
 * Set the max number of requests of brick of voxel loads.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbBrickLoads( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbBrickLoads( pValue );
}

/******************************************************************************
 * Set the request strategy indicating if, during data structure traversal,
 * priority of requests is set on brick loads or on node subdivisions first.
 *
 * @param pFlag the flag indicating the request strategy
 ******************************************************************************/
void SampleCore::setRendererPriorityOnBricks( bool pFlag )
{
	/*_pipeline->editRenderer()*/_renderer->setPriorityOnBricks( pFlag );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	/*_pipeline->editRenderer()*/_renderer->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::hasLight() const
{
	return false;
}

/******************************************************************************
 * Get the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::getLightPosition( float& pX, float& pY, float& pZ ) const
{
	pX = _lightPosition.x;
	pY = _lightPosition.y;
	pZ = _lightPosition.z;
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	// WARNING
	// Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
	// Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
	_lightPosition.x = pX/* - _translation[ 0 ]*/;
	_lightPosition.y = pY/* - _translation[ 1 ]*/;
	_lightPosition.z = pZ/* - _translation[ 2 ]*/;

	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &_lightPosition, sizeof( _lightPosition ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the translation used to position the GigaVoxels data structure
 *
 * @param pX the x componenet of the translation
 * @param pX the y componenet of the translation
 * @param pX the z componenet of the translation
 ******************************************************************************/
void SampleCore::getTranslation( float& pX, float& pY, float& pZ ) const
{
	pX = _translation[ 0 ];
	pY = _translation[ 1 ];
	pZ = _translation[ 2 ];
}

/******************************************************************************
 * Get the number of requests of node subdivisions the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbNodeSubdivisionRequests() const
{
	return _pipeline->getCache()->getNbNodeSubdivisionRequests();
}

/******************************************************************************
 * Get the number of requests of brick of voxel loads the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbBrickLoadRequests() const
{
	return _pipeline->getCache()->getNbBrickLoadRequests();
}

/******************************************************************************
 * Get the cache policy
 *
 * @return the cache policy
 ******************************************************************************/
unsigned int SampleCore::getCachePolicy() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getPolicy();
}

/******************************************************************************
 * Set the cache policy
 *
 * @param pValue the cache policy
 ******************************************************************************/
void SampleCore::setCachePolicy( unsigned int pValue )
{
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( static_cast< DataProductionManagerType::NodesCacheManager::ECachePolicy >( pValue ) );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( static_cast< DataProductionManagerType::BricksCacheManager::ECachePolicy >( pValue ) );
}

/******************************************************************************
 * Get the node cache memory
 *
 * @return the node cache memory
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheMemory() const
{
	return NODEPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the node cache memory
 *
 * @param pValue the node cache memory
 ******************************************************************************/
void SampleCore::setNodeCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache memory
 *
 * @return the brick cache memory
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheMemory() const
{
	return BRICKPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the brick cache memory
 *
 * @param pValue the brick cache memory
 ******************************************************************************/
void SampleCore::setBrickCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the node cache capacity
 *
 * @return the node cache capacity
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheCapacity() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the node cache capacity
 *
 * @param pValue the node cache capacity
 ******************************************************************************/
void SampleCore::setNodeCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache capacity
 *
 * @return the brick cache capacity
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheCapacity() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the brick cache capacity
 *
 * @param pValue the brick cache capacity
 ******************************************************************************/
void SampleCore::setBrickCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the number of unused nodes in cache
 *
 * @return the number of unused nodes in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedNodes() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Get the number of unused bricks in cache
 *
 * @return the number of unused bricks in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedBricks() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::has3DModel() const
{
	return true;
}

/******************************************************************************
 * Get the 3D model filename to load
 *
 * @return the 3D model filename to load
 ******************************************************************************/
string SampleCore::get3DModelFilename() const
{
	return _proxyGeometry->get3DModelFilename();
}

/******************************************************************************
 * Set the 3D model filename to load
 *
 * @param pFilename the 3D model filename to load
 ******************************************************************************/
void SampleCore::set3DModelFilename( const string& pFilename )
{
	// Store current proxy geometry state
	const bool screenBasedCriteria =_proxyGeometry->getScreenBasedCriteria();
	const float screenBasedCriteriaCoefficient =_proxyGeometry->getScreenBasedCriteriaCoefficient();
	const float materialAlphaCorrectionCoefficient =_proxyGeometry->getMaterialAlphaCorrectionCoefficient();
	
	// ---- Delete the 3D scene if needed ----
	
	if ( _proxyGeometry != NULL )
	{
		delete _proxyGeometry;
		_proxyGeometry = NULL;

		// Clear the GigaVoxels cache
		_pipeline->editCache()->clearCache();
	}

	// Initialize proxy geometry (load the 3D scene)
	//
	// - find a way to modify internal buffer size
	_proxyGeometry = new ProxyGeometry();
	_proxyGeometry->set3DModelFilename( pFilename );
	_proxyGeometry->initialize();
	// Restore previous proxy geometry state
	_proxyGeometry->setScreenBasedCriteria( screenBasedCriteria );
	_proxyGeometry->setScreenBasedCriteriaCoefficient( screenBasedCriteriaCoefficient );
	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( materialAlphaCorrectionCoefficient );
	/*_pipeline->editRenderer()*/_renderer->setProxyGeometry( _proxyGeometry );
	// Reset proxy geometry resources
	/*_pipeline->editRenderer()*/_renderer->unregisterProxyGeometryGraphicsResources();
	_proxyGeometry->setBufferSize( _width, _height );
	/*_pipeline->editRenderer()*/_renderer->registerProxyGeometryGraphicsResources();
	// Noise parameters
	setNoiseFirstFrequency( _noiseFirstFrequency );
	setNoiseStrength( _noiseStrength );
}

/******************************************************************************
 * Tell wheter or not the screen-based criteria is activated
 *
 * @return a flag telling wheter or not the screen-based criteria is activated
 ******************************************************************************/
bool SampleCore::getScreenBasedCriteria() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getScreenBasedCriteria();
}

/******************************************************************************
 * Set the flag telling wheter or not the screen-based criteria is activated
 *
 * @param pFlag a flag telling wheter or not the screen-based criteria is activated
 ******************************************************************************/
void SampleCore::setScreenBasedCriteria( bool pFlag )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setScreenBasedCriteria( pFlag );

	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenBasedCriteria), &pFlag, sizeof( pFlag ), 0, hipMemcpyHostToDevice ) );

	// Clear the GigaVoxels cache
	clearCache();
}

/******************************************************************************
 * Get the screen-based criteria coefficient
 *
 * @return the screen-based criteria coefficient
 ******************************************************************************/
float SampleCore::getScreenBasedCriteriaCoefficient() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getScreenBasedCriteriaCoefficient();
}

/******************************************************************************
 * Set the screen-based criteria coefficient
 *
 * @param pValue the screen-based criteria coefficient
 ******************************************************************************/
void SampleCore::setScreenBasedCriteriaCoefficient( float pValue )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setScreenBasedCriteriaCoefficient( pValue );

	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenBasedCriteriaCoefficient), &pValue, sizeof( pValue ), 0, hipMemcpyHostToDevice ) );

	// Clear the GigaVoxels cache
	clearCache();
}

/******************************************************************************
 * Get the material alpha correction coefficient
 *
 * @return the material alpha correction coefficient
 ******************************************************************************/
float SampleCore::getMaterialAlphaCorrectionCoefficient() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getMaterialAlphaCorrectionCoefficient();
}

/******************************************************************************
 * Set the material alpha correction coefficient
 *
 * @param pValue the material alpha correction coefficient
 ******************************************************************************/
void SampleCore::setMaterialAlphaCorrectionCoefficient( float pValue )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( pValue );

	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMaterialAlphaCorrectionCoefficient), &pValue, sizeof( pValue ), 0, hipMemcpyHostToDevice ) );

	// No need to clear the cache
	// - this variable is evaluated in the shader, not in the producer.
}

/******************************************************************************
 * Get the noise first frequency
 *
 * @return the noise first frequency
 ******************************************************************************/
float SampleCore::getNoiseFirstFrequency() const
{
	return _noiseFirstFrequency;
}

/******************************************************************************
 * Set the noise first frequency
 *
 * @param pValue the noise first frequency
 ******************************************************************************/
void SampleCore::setNoiseFirstFrequency( float pValue )
{
	_noiseFirstFrequency = pValue;
	
	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNoiseFirstFrequency), &_noiseFirstFrequency, sizeof( _noiseFirstFrequency ), 0, hipMemcpyHostToDevice ) );

	// Clear cache
	clearCache();
}

/******************************************************************************
 * Get the noise strength
 *
 * @return the noise strength
 ******************************************************************************/
float SampleCore::getNoiseStrength() const
{
	return _noiseStrength;
}

/******************************************************************************
 * Set the noise strength
 *
 * @param pValue the noise strength
 ******************************************************************************/
void SampleCore::setNoiseStrength( float pValue )
{
	_noiseStrength = pValue;
	
	// Update device memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNoiseStrength), &_noiseStrength, sizeof( _noiseStrength ), 0, hipMemcpyHostToDevice ) );

	// Clear cache
	clearCache();
}

/******************************************************************************
 * Get the voxel size multiplier
 *
 * @return the voxel size multiplier
 ******************************************************************************/
float SampleCore::getVoxelSizeMultiplier() const
{
	return /*_pipeline->editRenderer()*/_renderer->getVoxelSizeMultiplier();
}

/******************************************************************************
 * Set the voxel size multiplier
 *
 * @param the voxel size multiplier
 ******************************************************************************/
void SampleCore::setVoxelSizeMultiplier( float pValue )
{
	/*_pipeline->editRenderer()*/_renderer->setVoxelSizeMultiplier( pValue );
}

/******************************************************************************
 * Get the nodes cache usage
 *
 * @return the nodes cache usage
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheUsage() const
{
	//const unsigned int nbProducedElements = _pipeline->getCache()->getNodesCacheManager()->_totalNumLoads;
	const unsigned int nbProducedElements = _pipeline->getCache()->getNodesCacheManager()->_numElemsNotUsed;
	const unsigned int nbElements = _pipeline->getCache()->getNodesCacheManager()->getNumElements();

	const unsigned int cacheUsage = static_cast< unsigned int >( 100.0f * static_cast< float >( nbElements - nbProducedElements ) / static_cast< float >( nbElements ) );

	//std::cout << "NODE cache usage [ " << nbProducedElements << " / "<< nbElements << " : " << cacheUsage << std::endl;

	return cacheUsage;
}

/******************************************************************************
 * Get the bricks cache usage
 *
 * @return the bricks cache usage
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheUsage() const
{
	//const unsigned int nbProducedElements = _pipeline->getCache()->getBricksCacheManager()->_totalNumLoads;
	const unsigned int nbProducedElements = _pipeline->getCache()->getBricksCacheManager()->_numElemsNotUsed;
	const unsigned int nbElements = _pipeline->getCache()->getBricksCacheManager()->getNumElements();

	const unsigned int cacheUsage = static_cast< unsigned int >( 100.0f * static_cast< float >( nbElements - nbProducedElements ) / static_cast< float >( nbElements ) );

	//std::cout << "BRICK cache usage [ " << nbProducedElements << " / "<< nbElements << " : " << cacheUsage << std::endl;

	return cacheUsage;
}

/******************************************************************************
 * Get the number of tree leaf nodes
 *
 * @return the number of tree leaf nodes
 ******************************************************************************/
unsigned int SampleCore::getNbTreeLeafNodes() const
{
	return _pipeline->getCache()->_nbLeafNodes;
}

/******************************************************************************
 * Get the number of tree nodes
 *
 * @return the number of tree nodes
 ******************************************************************************/
unsigned int SampleCore::getNbTreeNodes() const
{
	return _pipeline->getCache()->_nbNodes;
}

/******************************************************************************
* Get the flag indicating wheter or not data production monitoring is activated
*
* @return the flag indicating wheter or not data production monitoring is activated
 ******************************************************************************/
bool SampleCore::hasDataProductionMonitoring() const
{
	return true;
}

/******************************************************************************
* Set the the flag indicating wheter or not data production monitoring is activated
*
* @param pFlag the flag indicating wheter or not data production monitoring is activated
 ******************************************************************************/
void SampleCore::setDataProductionMonitoring( bool pFlag )
{
}

/******************************************************************************
* Get the flag indicating wheter or not cache monitoring is activated
*
* @return the flag indicating wheter or not cache monitoring is activated
 ******************************************************************************/
bool SampleCore::hasCacheMonitoring() const
{
	return true;
}

/******************************************************************************
* Set the the flag indicating wheter or not cache monitoring is activated
*
* @param pFlag the flag indicating wheter or not cache monitoring is activated
 ******************************************************************************/
void SampleCore::setCacheMonitoring( bool pFlag )
{
}

/******************************************************************************
* Get the flag indicating wheter or not time budget monitoring is activated
*
* @return the flag indicating wheter or not time budget monitoring is activated
 ******************************************************************************/
bool SampleCore::hasTimeBudgetMonitoring() const
{
	return true;
}

/******************************************************************************
* Set the the flag indicating wheter or not time budget monitoring is activated
*
* @param pFlag the flag indicating wheter or not time budget monitoring is activated
 ******************************************************************************/
void SampleCore::setTimeBudgetMonitoring( bool pFlag )
{
}

/******************************************************************************
 *Tell wheter or not time budget is acivated
 *
 * @return a flag to tell wheter or not time budget is activated
 ******************************************************************************/
bool SampleCore::hasRenderingTimeBudget() const
{
	return true;
}

/******************************************************************************
 * Set the flag telling wheter or not time budget is acivated
 *
 * @param pFlag a flag to tell wheter or not time budget is activated
 ******************************************************************************/
void SampleCore::setRenderingTimeBudgetActivated( bool pFlag )
{
}

/******************************************************************************
 * Get the user requested time budget
 *
 * @return the user requested time budget
 ******************************************************************************/
unsigned int SampleCore::getRenderingTimeBudget() const
{
	return static_cast< unsigned int >( /*_pipeline->getRenderer()*/_renderer->getTimeBudget() );
}

/******************************************************************************
 * Set the user requested time budget
 *
 * @param pValue the user requested time budget
 ******************************************************************************/
void SampleCore::setRenderingTimeBudget( unsigned int pValue )
{
	/*_pipeline->editRenderer()*/_renderer->setTimeBudget( static_cast< float >( pValue ) );
}

/******************************************************************************
 * This method return the duration of the timer event between start and stop event
 *
 * @return the duration of the event in milliseconds
 ******************************************************************************/
float SampleCore::getRendererElapsedTime() const
{
	return /*_pipeline->editRenderer()*/_renderer->getElapsedTime();
}

/******************************************************************************
 * Tell wheter or not pipeline uses programmable shaders
 *
 * @return a flag telling wheter or not pipeline uses programmable shaders
 ******************************************************************************/
bool SampleCore::hasProgrammableShaders() const
{
	return false;//true;
}

/******************************************************************************
 * Tell wheter or not pipeline has a given type of shader
 *
 * @param pShaderType the type of shader to test
 *
 * @return a flag telling wheter or not pipeline has a given type of shader
 ******************************************************************************/
bool SampleCore::hasShaderType( unsigned int pShaderType ) const
{
	return false;//_shaderProgram->hasShaderType( static_cast< GsShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * Get the source code associated to a given type of shader
 *
 * @param pShaderType the type of shader
 *
 * @return the associated shader source code
 ******************************************************************************/
std::string SampleCore::getShaderSourceCode( unsigned int pShaderType ) const
{
	return string();//_shaderProgram->getShaderSourceCode( static_cast< GsShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * Get the filename associated to a given type of shader
 *
 * @param pShaderType the type of shader
 *
 * @return the associated shader filename
 ******************************************************************************/
std::string SampleCore::getShaderFilename( unsigned int pShaderType ) const
{
	return string();// _shaderProgram->getShaderFilename( static_cast< GsShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * ...
 *
 * @param pShaderType the type of shader
 *
 * @return ...
 ******************************************************************************/
bool SampleCore::reloadShader( unsigned int pShaderType )
{
	return false;//_shaderProgram->reloadShader( static_cast< GsShaderProgram::ShaderType >( pShaderType ) );
}
