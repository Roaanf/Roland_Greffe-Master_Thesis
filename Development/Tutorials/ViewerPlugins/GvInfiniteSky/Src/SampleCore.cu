#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/GsVector.h>
#include <GvStructure/GsVolumeTree.h>
#include <GvStructure/GsDataProductionManager.h>
#include <GvUtils/GsSimplePipeline.h>
#include <GvUtils/GsSimplePriorityPoliciesManagerKernel.h>
#include <GvCore/GsError.h>
#include <GvPerfMon/GsPerformanceMonitor.h>

// Project
#include "Producer.h"
#include "Shader.h"
#include "VolumeTreeRendererCUDA.h"
#include "CustomEditor.h"

// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>
#include <GvvPipelineInterfaceViewer.h>
#include <GvvPluginInterface.h>
#include <GvvPluginManager.h>
#include <Gvv3DWindow.h>
#include <GvvPipelineManager.h>
#include <GvvEditorWindow.h>
#include <GvvPipelineEditor.h>

// Cuda SDK
#include <hip/hip_vector_types.h>

// System
#include <cstdlib>
#include <ctime>
#include <cassert>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;

// GigaVoxels viewer
using namespace GvViewerCore;


/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

/**
 * Defines the size allowed for each type of pool
 */
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 * all the different rotation matrices
 ******************************************************************************/

 float Id[9] = {1,0,0,0,1,0,0,0,1}; //inv Id
 float R0[9] = {0,-1,0,1,0,0,0,0,1}; // inv R1
 float R1[9] = {0,1,0,-1,0,0,0,0,1}; // inv R0
 float R2[9] = {1,0,0,0,0,-1,0,1,0};// inv R3
 float R3[9] = {1,0,0,0,0,1,0,-1,0}; // inv R2
 float R4[9] = {0,0,1,0,1,0,-1,0,0};// inv R5
 float R5[9] = {0,0,-1,0,1,0,1,0,0}; // inv R4
 float R6[9] = {0,0,1,1,0,0,0,1,0}; // inv R13
 float R7[9] = {0,0,-1,-1,0,0,0,1,0}; // inv R12
 float R8[9] = {0,-1,0,0,0,-1,1,0,0}; // inv R10
 float R9[9] = {0,1,0,0,0,-1,-1,0,0}; // inv R11
 float R10[9] = {0,0,1,-1,0,0,0,-1,0}; // inv R8
 float R11[9] = {0,0,-1,1,0,0,0,-1,0}; // inv R9
 float R12[9] = {0,-1,0,0,0,1,-1,0,0}; // inv R7
 float R13[9] = {0,1,0,0,0,1,1,0,0};  // inv R6
 float R14[9] = {1,0,0,0,-1,0,0,0,-1}; // inv R14
 float R15[9] = {-1,0,0,0,1,0,0,0,-1}; // inv R15
 float R16[9] = {-1,0,0,0,-1,0,0,0,1}; // inv R16
 float R17[9] = {0,0,1,0,-1,0,1,0,0}; // inv R17
 float R18[9] = {0,0,-1,0,-1,0,-1,0,0}; // inv R18
 float R19[9] = {0,-1,0,-1,0,0,0,0,-1};  // inv R19
 float R20[9] = {0,1,0,1,0,0,0,0,-1}; // inv R20
 float R21[9] = {-1,0,0,0,0,-1,0,-1,0};  // inv R21
 float R22[9] = {-1,0,0,0,0,1,0,1,0}; // inv R22


/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	GvvPipelineInterface()
,	_pipeline( NULL )
,	_volumeTree( NULL )
,	_cache( NULL )
,	_renderer( NULL )
,	_producer( NULL )
,	_colorTex( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_depthBuffer( 0 )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 0 )
{
	// Translation used to position the GigaVoxels data structure
	_translation[ 0 ] = -0.5f;
	_translation[ 1 ] = -0.5f;
	_translation[ 2 ] = -0.5f;

	// Light position
	_lightPosition = make_float3( 1.f, 1.f, 1.f );

	// Spheres ray-tracing parameters
	_nbSpheres = 0;
	_userDefinedMinLevelOfResolutionToHandleMode = false;
    _userDefinedMinLevelOfResolutionToHandle = 0;
	_automaticMinLevelOfResolutionToHandleMode = true;
	//_automaticMinLevelOfResolutionToHandle = 0;
	_sphereBrickIntersectionType = 0;
	_geometricCriteria = true;
	_minNbSpheresPerBrick = 1;
	_screenBasedCriteria = true;
	_absoluteSizeCriteria = true;
	_fixedSizeSphere = true;
	_meanSizeOfSpheres = false;
	_shaderUseUniformColor = false;
	_shaderUniformColor = make_float4( 1.f, 1.f, 1.f, 1.f );
    _shaderAnimation = false;
    _shaderBlurSphere = false;
    _shaderFog = false;
    _shaderFogDensity = 0.0f;
    _shading = false;
    _bugCorrection = true;

	// Infinite sky navigation parameters
    _sphereDiameterCoeff = 1;
	cameraInBrick = make_float3(0.5f,0.5f,0.5f);
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNbCameraReflections), &cameraInBrick, sizeof( cameraInBrick ), 0, hipMemcpyHostToDevice ) );
	_numberOfReflections = 0;
	matrix =Id;
	antiMatrix = Id;
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "InfiniteSky";
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
    printf( "*********** SAMPLE CORE INIT() ***********\n" );
	CUDAPM_INIT();

	// Pipeline creation
	_pipeline = new PipelineType();
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, shader );

	// Producer initialization
	_producer = new ProducerType();
	assert( _producer != NULL );
	_pipeline->addProducer( _producer );

	// Renderer initialization
	_renderer = new RendererType( _pipeline->editDataStructure(), _pipeline->editCache() );
	assert( _renderer != NULL );
	_pipeline->addRenderer( _renderer );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );

	// Retrieve GigaSpace's main objects to ease user development
	_volumeTree = _pipeline->editDataStructure();
	_cache = _pipeline->editCache();
		
	// Custom initialization
	// Note : this could be done via an XML settings file loaded at initialization
	setNbSpheres( 1 );
    setNbSpheresTotal ( 1 );
	// Sphere-brick intersection type
	//
	// 0 : sphere-sphere (brick are approximated by spheres)
	// 1 : sphere-box (brick are not approximated, it uses real sphere-box intersection test)
	setSphereBrickIntersectionType( 1 );
    setSphereRadiusFader( 1.0f );
	setGeometricCriteria( true );
	setMinNbSpheresPerBrick( 1 );
	setScreenBasedCriteria( true );
	setAbsoluteSizeCriteria( true );
    setSphereDiameterCoeff( 1 );
	setFixedSizeSphere( true );
	setFixedSizeSphereRadius( 0.01f );
	setMeanSizeOfSpheres( false );
	setShaderUniformColorMode( false );
	setShaderUniformColor( 1.f, 1.f, 1.f, 1.f );
    setShaderAnimation( false );
    setShaderFog(false);
    setShaderFogColor( 0.8f, .8f, .8f, 1.f);
    setFogDensity(.15f);
    setIlluminationCoeff(1.0f);
	
	// Configure cache
	_cache->setMaxNbNodeSubdivisions( 500 );
	_cache->setMaxNbBrickLoads( 300 );
	_cache->editNodesCacheManager()->setPolicy( DataProductionManager::NodesCacheManager::eAllPolicies );
	_cache->editBricksCacheManager()->setPolicy(DataProductionManager::BricksCacheManager::eAllPolicies );

	// Set light position
	setLightPosition( 1.f, 1.f, 1.f );

	qglviewer::Vec position;
	position.x = 0.5;
	position.y = 0.5;
	position.z = 0.5;
}

/******************************************************************************
 * setter of the pipeline viewer containing the camera
 *
 * @param pPipelineViewer the pipeline viewer to be set
 ******************************************************************************/
void SampleCore::setPipelineViewer( GvViewerGui::GvvPipelineInterfaceViewer* pPipelineViewer ) 
{
	_pipelineViewer = pPipelineViewer;
}

/******************************************************************************
 * multiply a point by a rotation matrix to rotate it around (0.5;0.5;0.5)
 *
 * @param mat the rotation matrix
 * @param x the point to rotate around (0.5;0.5;0.5) 
 ******************************************************************************/
void multMatrix( float* mat, qglviewer::Vec* x )
{
	float xx,yy,zz;

	xx= (x->x) - 0.5f;
	yy= (x->y) - 0.5f;
	zz= (x->z) - 0.5f;
	float xxx,yyy,zzz;

	xxx = mat[0]*(xx)+mat[1]*(yy)+mat[2]*(zz);
	yyy = mat[3]*(xx)+mat[4]*(yy)+mat[5]*(zz);
	zzz = mat[6]*(xx)+mat[7]*(yy)+mat[8]*(zz);

	x->x = xxx+0.5f;
	x->y = yyy+0.5f;
	x->z = zzz+0.5f;
}

/******************************************************************************
 * multiply a vector by a rotation matrix 
 *
 * @param mat the rotation matrix
 * @param x the vector to rotate
 ******************************************************************************/
void multMatrix2( float* mat, qglviewer::Vec* x )
{
	float xx,yy,zz;
	xx = mat[0]*(x->x)+mat[1]*(x->y)+mat[2]*(x->z);
	yy = mat[3]*(x->x)+mat[4]*(x->y)+mat[5]*(x->z);
	zz = mat[6]*(x->x)+mat[7]*(x->y)+mat[8]*(x->z);

	x->x = xx;
	x->y = yy;
	x->z = zz;
}


/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{

	//_pipelineViewer->camera()->setFlySpeed( 0.04 );
	qglviewer::Vec position =_pipelineViewer->camera()->position();
	
	//float zNear = _pipelineViewer->camera()->upd
	//float epsilon = 1e-4;
	
	bool reflection = false;
	
	if ( position.x > 1 )
	{
		/*orientation.x = - orientation.x;
		upVector.x = -upVector.x;
		position.x=2-position.x-2*epsilon;*/

		position.x = 0+ position.x - 1;

		qglviewer::Vec centre_face (1.f,0.5f,0.5f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
				
		reflection = true;
	}
	if ( position.x < 0  )
	{
		/*orientation.x = - orientation.x;
		upVector.x = -upVector.x;
		position.x=-position.x+2*epsilon;*/
		
		position.x = 1 - (-position.x);
		qglviewer::Vec centre_face (0.f,0.5f,0.5f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
				
		reflection = true;
	}
	if ( position.y > 1)
	{
		/*orientation.y = - orientation.y;
		upVector.y = -upVector.y;
		position.y=2-position.y-2*epsilon;*/

		position.y = 0+ position.y - 1;
		qglviewer::Vec centre_face (0.5f,1.f,0.5f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
			
		reflection = true;
	}
	if ( position.y < 0  )
	{
		/*orientation.y = - orientation.y;
		upVector.y = -upVector.y;
		position.y=-position.y+2*epsilon;*/

		position.y = 1 - (-position.y);
		qglviewer::Vec centre_face (0.5f,0.f,0.5f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
			
		reflection = true;
	}
	if ( position.z > 1 )
	{
		/*orientation.z = - orientation.z;
		upVector.z = -upVector.z;
		position.z=2-position.z-2*epsilon;*/

		position.z = 0+ position.z - 1;
		qglviewer::Vec centre_face (0.5f,0.5f,1.f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
			
		reflection = true;
	}
	if ( position.z < 0 )
	{
		/*orientation.z = - orientation.z;
		upVector.z = -upVector.z;
		position.z=-position.z+2*epsilon;*/

		position.z = 1 - (-position.z);
		qglviewer::Vec centre_face (0.5f,0.5f,0.f);
		multMatrix(antiMatrix,&centre_face);

		cameraInBrick += make_float3((centre_face.x - 0.5)*2 , (centre_face.y - 0.5)*2,(centre_face.z - 0.5)*2 );
			
		reflection = true;
	}

	if ( reflection )
	{
		qglviewer::Vec orientation = _pipelineViewer->camera()->viewDirection();
		qglviewer::Vec upVector = _pipelineViewer->camera()->upVector();
		//printf("in :  %f,%f,%f\n",rayStartTree.x,rayStartTree.y,rayStartTree.z);

		multMatrix(antiMatrix,&position);		
		multMatrix2(antiMatrix,&orientation);
		multMatrix2(antiMatrix,&upVector);

		
		switch (  abs((  (int)((cameraInBrick.x-0.5)*2 + (cameraInBrick.y-0.5)*3 + (cameraInBrick.z-0.5)*5   )) % 24))
		{

			case 0 : matrix = Id; antiMatrix = Id ; break;
			case 1 : matrix = R0; antiMatrix = R1 ;  break;
			case 2 : matrix = R1; antiMatrix = R0 ; break;
			case 3 : matrix = R2; antiMatrix = R3 ; break;
			case 4 : matrix = R3; antiMatrix = R2 ; break;
			case 5 : matrix = R4; antiMatrix = R5 ; break;
			case 6 : matrix = R5; antiMatrix = R4 ; break;
			case 7 : matrix = R6; antiMatrix = R13 ; break;
			case 8 : matrix = R7; antiMatrix = R12 ; break;
			case 9 : matrix = R8; antiMatrix = R10 ; break;
			case 10 : matrix = R9; antiMatrix = R11 ; break;
			case 11 : matrix = R10; antiMatrix = R8 ; break;
			case 12 : matrix = R11; antiMatrix = R9 ; break;
			case 13 : matrix = R12; antiMatrix = R7 ; break;
			case 14 : matrix = R13; antiMatrix = R6 ; break;
			case 15 : matrix = R14; antiMatrix = R14 ; break;
			case 16 : matrix = R15; antiMatrix = R15 ; break;
			case 17 : matrix = R16; antiMatrix = R16 ; break;
			case 18 : matrix = R17; antiMatrix = R17 ; break;
			case 19 : matrix = R18; antiMatrix = R18 ; break;
			case 20 : matrix = R19; antiMatrix = R19 ; break;
			case 21 : matrix = R20; antiMatrix = R20 ; break;
			case 22 : matrix = R21; antiMatrix = R21 ; break;
			case 23 : matrix = R22; antiMatrix = R22 ; break;
			
			
			default :              break;
		}
		multMatrix(matrix,&position);		
		multMatrix2(matrix,&orientation);
		multMatrix2(matrix,&upVector);
		//printf("%f,%f,%f\n",cameraInBrick.x,cameraInBrick.y,cameraInBrick.z);

		//_numberOfReflections+=reflection;
	

		GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNbCameraReflections), &cameraInBrick, sizeof( cameraInBrick ), 0, hipMemcpyHostToDevice ) );
		
	
		_pipelineViewer->camera()->setPosition(position);
		_pipelineViewer->camera()->setViewDirection(orientation);
		_pipelineViewer->camera()->setUpVector(upVector);
		_pipelineViewer->camera()->loadModelViewMatrix();
	}
	
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

    uchar4 color = _renderer->getClearColor();
    glClearColor( color.x/255.f, color.y/255.f, color.z/255.f, color.w/255.f );

    glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );

	glMatrixMode( GL_MODELVIEW );

	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		// Translation used to position the GigaVoxels data structure
		//glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
		_volumeTree->render();
		glPopMatrix();
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
		glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		glClear( GL_COLOR_BUFFER_BIT );
	}

	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

	// extract viewport
	GLint params[4];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4(params[0], params[1], params[2], params[3]);

	// render the scene into textures
	CUDAPM_STOP_EVENT( app_init_frame );

	// Build the world transformation matrix
	float4x4 modelMatrix;
	glPushMatrix();
	glLoadIdentity();
	// Translation used to position the GigaVoxels data structure
	//glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// Render
	_pipeline->execute( modelMatrix, viewMatrix, projectionMatrix, viewport );

	// Render the result to the screen
	glMatrixMode( GL_MODELVIEW );
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode( GL_PROJECTION );
	glPushMatrix();
	glLoadIdentity();

	glDisable( GL_DEPTH_TEST );
	glEnable( GL_TEXTURE_RECTANGLE_EXT );
	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );

	// Draw a full screen quad
	GLint	sMin = 0;
	GLint	tMin = 0;
	GLint	sMax = _width;
	GLint	tMax = _height;


	glBegin( GL_QUADS );
	glColor3f( 1.0f, 1.0f, 1.0f );
	glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
	glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
	glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
	glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
	glEnd();

	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );
	glDisable( GL_TEXTURE_RECTANGLE_EXT );

	glPopMatrix();
	glMatrixMode( GL_MODELVIEW );
	glPopMatrix();

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	//_renderer->doPostRender();

	// Update GigaVoxels info
	_renderer->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the GigaVoxels performance monitor (if it has been activated during GigaVoxels compilation)
	if ( _displayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL( _displayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	_width = width;
	_height = height;

	// Reset default active frame region for rendering
	_renderer->setProjectedBBox( make_uint4( 0, 0, _width, _height ) );

	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( _width, _height ) );

	// Create frame-dependent objects
	
	// Disconnect all registered graphics resources
	_renderer->resetGraphicsResources();
	
	// ...
	if (_depthBuffer)
	{
		glDeleteBuffers(1, &_depthBuffer);
	}

	if (_colorTex)
	{
		glDeleteTextures(1, &_colorTex);
	}
	if (_depthTex)
	{
		glDeleteTextures(1, &_depthTex);
	}

	if (_frameBuffer)
	{
		glDeleteFramebuffers(1, &_frameBuffer);
	}

	glGenTextures(1, &_colorTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, _colorTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenBuffers(1, &_depthBuffer);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, _depthBuffer);
	glBufferData(GL_PIXEL_PACK_BUFFER, width * height * sizeof(GLuint), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glGenTextures(1, &_depthTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, _depthTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_DEPTH24_STENCIL8_EXT, width, height, 0, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenFramebuffers( 1, &_frameBuffer );
	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_RECTANGLE_EXT, _colorTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, _depthTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_STENCIL_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, _depthTex, 0 );
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );
	GV_CHECK_GL_ERROR();

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	_renderer->resetGraphicsResources();

	if ( _displayOctree )
	{
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_renderer->connect( GsGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Get the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::getDataStructureAppearance( bool& pShowNodeHasBrickTerminal, bool& pShowNodeHasBrickNotTerminal, bool& pShowNodeIsBrickNotInCache, bool& pShowNodeEmptyOrConstant
											, float& pNodeHasBrickTerminalColorR, float& pNodeHasBrickTerminalColorG, float& pNodeHasBrickTerminalColorB, float& pNodeHasBrickTerminalColorA
											, float& pNodeHasBrickNotTerminalColorR, float& pNodeHasBrickNotTerminalColorG, float& pNodeHasBrickNotTerminalColorB, float& pNodeHasBrickNotTerminalColorA
											, float& pNodeIsBrickNotInCacheColorR, float& pNodeIsBrickNotInCacheColorG, float& pNodeIsBrickNotInCacheColorB, float& pNodeIsBrickNotInCacheColorA
											, float& pNodeEmptyOrConstantColorR, float& pNodeEmptyOrConstantColorG, float& pNodeEmptyOrConstantColorB, float& pNodeEmptyOrConstantColorA ) const
{
	float4 nodeHasBrickTerminalColor;
	float4 nodeHasBrickNotTerminalColor;
	float4 nodeIsBrickNotInCacheColor;
	float4 nodeEmptyOrConstantColor;
										
	_volumeTree->getDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );

	pNodeHasBrickTerminalColorR = nodeHasBrickTerminalColor.x;
	pNodeHasBrickTerminalColorG = nodeHasBrickTerminalColor.y;
	pNodeHasBrickTerminalColorB = nodeHasBrickTerminalColor.z;
	pNodeHasBrickTerminalColorA = nodeHasBrickTerminalColor.w;

	pNodeHasBrickNotTerminalColorR = nodeHasBrickNotTerminalColor.x;
	pNodeHasBrickNotTerminalColorG = nodeHasBrickNotTerminalColor.y;
	pNodeHasBrickNotTerminalColorB = nodeHasBrickNotTerminalColor.z;
	pNodeHasBrickNotTerminalColorA = nodeHasBrickNotTerminalColor.w;

	pNodeIsBrickNotInCacheColorR = nodeIsBrickNotInCacheColor.x;
	pNodeIsBrickNotInCacheColorG = nodeIsBrickNotInCacheColor.y;
	pNodeIsBrickNotInCacheColorB = nodeIsBrickNotInCacheColor.z;
	pNodeIsBrickNotInCacheColorA = nodeIsBrickNotInCacheColor.w;

	pNodeEmptyOrConstantColorR = nodeEmptyOrConstantColor.x;
	pNodeEmptyOrConstantColorG = nodeEmptyOrConstantColor.y;
	pNodeEmptyOrConstantColorB = nodeEmptyOrConstantColor.z;
	pNodeEmptyOrConstantColorA = nodeEmptyOrConstantColor.w;
}

/******************************************************************************
 * Set the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::setDataStructureAppearance( bool pShowNodeHasBrickTerminal, bool pShowNodeHasBrickNotTerminal, bool pShowNodeIsBrickNotInCache, bool pShowNodeEmptyOrConstant
											, float pNodeHasBrickTerminalColorR, float pNodeHasBrickTerminalColorG, float pNodeHasBrickTerminalColorB, float pNodeHasBrickTerminalColorA
											, float pNodeHasBrickNotTerminalColorR, float pNodeHasBrickNotTerminalColorG, float pNodeHasBrickNotTerminalColorB, float pNodeHasBrickNotTerminalColorA
											, float pNodeIsBrickNotInCacheColorR, float pNodeIsBrickNotInCacheColorG, float pNodeIsBrickNotInCacheColorB, float pNodeIsBrickNotInCacheColorA
											, float pNodeEmptyOrConstantColorR, float pNodeEmptyOrConstantColorG, float pNodeEmptyOrConstantColorB, float pNodeEmptyOrConstantColorA )
{
	float4 nodeHasBrickTerminalColor = make_float4( pNodeHasBrickTerminalColorR, pNodeHasBrickTerminalColorG, pNodeHasBrickTerminalColorB, pNodeHasBrickTerminalColorA );
	float4 nodeHasBrickNotTerminalColor = make_float4( pNodeHasBrickNotTerminalColorR, pNodeHasBrickNotTerminalColorG, pNodeHasBrickNotTerminalColorB, pNodeHasBrickNotTerminalColorA );
	float4 nodeIsBrickNotInCacheColor = make_float4( pNodeIsBrickNotInCacheColorR, pNodeIsBrickNotInCacheColorG, pNodeIsBrickNotInCacheColorB, pNodeIsBrickNotInCacheColorA );
	float4 nodeEmptyOrConstantColor = make_float4( pNodeEmptyOrConstantColorR, pNodeEmptyOrConstantColorG, pNodeEmptyOrConstantColorB, pNodeEmptyOrConstantColorA );

	_volumeTree->setDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_volumeTree->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_volumeTree->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Get the node tile resolution of the data structure.
 *
 * @param pX the X node tile resolution
 * @param pY the Y node tile resolution
 * @param pZ the Z node tile resolution
 ******************************************************************************/
void SampleCore::getDataStructureNodeTileResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& nodeTileResolution = _volumeTree->getNodeTileResolution().get();

	pX = nodeTileResolution.x;
	pY = nodeTileResolution.y;
	pZ = nodeTileResolution.z;
}

/******************************************************************************
 * Get the brick resolution of the data structure (voxels).
 *
 * @param pX the X brick resolution
 * @param pY the Y brick resolution
 * @param pZ the Z brick resolution
 ******************************************************************************/
void SampleCore::getDataStructureBrickResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& brickResolution = _volumeTree->getBrickResolution().get();

	pX = brickResolution.x;
	pY = brickResolution.y;
	pZ = brickResolution.z;
}

/******************************************************************************
 * Get the max depth.
 *
 * @return the max depth
 ******************************************************************************/
unsigned int SampleCore::getRendererMaxDepth() const
{
	return _volumeTree->getMaxDepth();
}

/******************************************************************************
 * Set the max depth.
 *
 * @param pValue the max depth
 ******************************************************************************/
void SampleCore::setRendererMaxDepth( unsigned int pValue )
{
	_volumeTree->setMaxDepth( pValue );
}

/******************************************************************************
 * Get the max number of requests of node subdivisions.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbNodeSubdivisions() const
{
	return _cache->getMaxNbNodeSubdivisions();
}

/******************************************************************************
 * Set the max number of requests of node subdivisions.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbNodeSubdivisions( unsigned int pValue )
{
	_cache->setMaxNbNodeSubdivisions( pValue );
}

/******************************************************************************
 * Get the max number of requests of brick of voxel loads.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbBrickLoads() const
{
	return _cache->getMaxNbBrickLoads();
}

/******************************************************************************
 * Set the max number of requests of brick of voxel loads.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbBrickLoads( unsigned int pValue )
{
	_cache->setMaxNbBrickLoads( pValue );
}

/******************************************************************************
 * Set the request strategy indicating if, during data structure traversal,
 * priority of requests is set on brick loads or on node subdivisions first.
 *
 * @param pFlag the flag indicating the request strategy
 ******************************************************************************/
void SampleCore::setRendererPriorityOnBricks( bool pFlag )
{
	_renderer->setPriorityOnBricks( pFlag );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	_renderer->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::hasLight() const
{
	return false;
}

/******************************************************************************
 * Get the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::getLightPosition( float& pX, float& pY, float& pZ ) const
{
	pX = _lightPosition.x;
	pY = _lightPosition.y;
	pZ = _lightPosition.z;
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
    // Update DEVICE memory with "light position"
    //
    // WARNING
    // Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
    // Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
    _lightPosition.x = pX/* - _translation[ 0 ]*/;
    _lightPosition.y = pY/* - _translation[ 1 ]*/;
    _lightPosition.z = pZ/* - _translation[ 2 ]*/;

    // Update device memory
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &_lightPosition, sizeof( _lightPosition ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the translation used to position the GigaVoxels data structure
 *
 * @param pX the x componenet of the translation
 * @param pX the y componenet of the translation
 * @param pX the z componenet of the translation
 ******************************************************************************/
void SampleCore::getTranslation( float& pX, float& pY, float& pZ ) const
{
	pX = _translation[ 0 ];
	pY = _translation[ 1 ];
	pZ = _translation[ 2 ];
}

/******************************************************************************
 * Get the number of requests of node subdivisions the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbNodeSubdivisionRequests() const
{
	return _cache->getNbNodeSubdivisionRequests();
}

/******************************************************************************
 * Get the number of requests of brick of voxel loads the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbBrickLoadRequests() const
{
	return _cache->getNbBrickLoadRequests();
}

/******************************************************************************
 * Get the cache policy
 *
 * @return the cache policy
 ******************************************************************************/
unsigned int SampleCore::getCachePolicy() const
{
	return _cache->getBricksCacheManager()->getPolicy();
}

/******************************************************************************
 * Set the cache policy
 *
 * @param pValue the cache policy
 ******************************************************************************/
void SampleCore::setCachePolicy( unsigned int pValue )
{
	_cache->editNodesCacheManager()->setPolicy( static_cast< DataProductionManager::NodesCacheManager::ECachePolicy>( pValue ) );
	_cache->editBricksCacheManager()->setPolicy( static_cast< DataProductionManager::BricksCacheManager::ECachePolicy>( pValue ) );
}

/******************************************************************************
 * Get the node cache memory
 *
 * @return the node cache memory
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheMemory() const
{
	return NODEPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the node cache memory
 *
 * @param pValue the node cache memory
 ******************************************************************************/
void SampleCore::setNodeCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache memory
 *
 * @return the brick cache memory
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheMemory() const
{
	return BRICKPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the brick cache memory
 *
 * @param pValue the brick cache memory
 ******************************************************************************/
void SampleCore::setBrickCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the node cache capacity
 *
 * @return the node cache capacity
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheCapacity() const
{
	return _cache->getNodesCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the node cache capacity
 *
 * @param pValue the node cache capacity
 ******************************************************************************/
void SampleCore::setNodeCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache capacity
 *
 * @return the brick cache capacity
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheCapacity() const
{
	return _cache->getBricksCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the brick cache capacity
 *
 * @param pValue the brick cache capacity
 ******************************************************************************/
void SampleCore::setBrickCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the number of unused nodes in cache
 *
 * @return the number of unused nodes in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedNodes() const
{
	return _cache->getNodesCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Get the number of unused bricks in cache
 *
 * @return the number of unused bricks in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedBricks() const
{
	return _cache->getBricksCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getNbSpheres() const
{
    return _nbSpheres;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setNbSpheres( unsigned int pValue )
{
    _nbSpheres = pValue;

    // Update producer
    _producer->setNbSpheres( pValue );

    // Update DEVICE memory
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNbSpheres), &_nbSpheres, sizeof( _nbSpheres ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getNbSpheresTotal() const
{
    return _nbSpheresTotal;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setNbSpheresTotal( unsigned int pValue )
{
    _nbSpheresTotal = pValue;

    unsigned int nbSphereByBrick = 0;

    // calcul du niveau de resolution necessaire a la production des etoiles
    unsigned int levelToHandle = 1 + logf(_nbSpheresTotal / 998) / log( 8.f );

    // calcul pour savoir combien il faut de spheres par brique
    nbSphereByBrick = _nbSpheresTotal / powf( 8, levelToHandle );

    // Update producer
    _producer->setNbSpheres( nbSphereByBrick );

    setUserDefinedMinLevelOfResolutionToHandle( levelToHandle );

    setRendererMaxDepth( levelToHandle );

    //printf("level To handle : %d\nnbSphere by brick : %d\n", levelToHandle, nbSphereByBrick);

    // Update DEVICE memory
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNbSpheres), &nbSphereByBrick, sizeof( nbSphereByBrick ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::regeneratePositions(){


    _producer->generateNewParticleBuffer();
    // Clear the cache
    clearCache();

}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::getUserDefinedMinLevelOfResolutionToHandleMode() const
{
	return _userDefinedMinLevelOfResolutionToHandleMode;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setUserDefinedMinLevelOfResolutionToHandleMode( bool pFlag )
{
	_userDefinedMinLevelOfResolutionToHandleMode = pFlag;

	// Update DEVICE memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMinLevelOfResolutionToHandle), &_userDefinedMinLevelOfResolutionToHandle, sizeof( _userDefinedMinLevelOfResolutionToHandle ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getUserDefinedMinLevelOfResolutionToHandle() const
{
	return _userDefinedMinLevelOfResolutionToHandle;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setUserDefinedMinLevelOfResolutionToHandle( unsigned int pValue )
{
	_userDefinedMinLevelOfResolutionToHandle = pValue;

	// Update DEVICE memory
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMinLevelOfResolutionToHandle), &_userDefinedMinLevelOfResolutionToHandle, sizeof( _userDefinedMinLevelOfResolutionToHandle ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::getAutomaticMinLevelOfResolutionToHandleMode() const
{
	return _automaticMinLevelOfResolutionToHandleMode;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setAutomaticMinLevelOfResolutionToHandleMode( bool pFlag )
{
	_automaticMinLevelOfResolutionToHandleMode = pFlag;

	if ( pFlag )
	{
		unsigned int minLevelOfResolutionToHandle = 0;
		const unsigned int nbChildren = 8;	// for octree
		const unsigned int maxNbSphereByBrick = 998;		// 10 x 10 x 10 - 2		[ 2 first cache elements are used to write special data ]
		unsigned int minNbSphereByBrick = static_cast< unsigned int >( static_cast< float >( _nbSpheres ) / powf( nbChildren, minLevelOfResolutionToHandle ) );
		while ( minNbSphereByBrick > maxNbSphereByBrick )
		{
			minLevelOfResolutionToHandle++;

			minNbSphereByBrick /= powf( nbChildren, minLevelOfResolutionToHandle );
		}

		// Update DEVICE memory
		GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMinLevelOfResolutionToHandle), &minLevelOfResolutionToHandle, sizeof( minLevelOfResolutionToHandle ), 0, hipMemcpyHostToDevice ) );

		// Clear the cache
		clearCache();
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getAutomaticMinLevelOfResolutionToHandle() const
{
	//unsigned int minLevelOfResolutionToHandle = 0;
	//const unsigned int nbChildren = 8;	// for octree
	//const unsigned int maxNbSphereByBrick = 998;		// 10 x 10 x 10 - 2		[ 2 first cache elements are used to write special data ]
	//unsigned int minNbSphereByBrick = static_cast< unsigned int >( static_cast< float >( _nbSpheres ) / powf( nbChildren, minLevelOfResolutionToHandle ) );
	//while ( minNbSphereByBrick > maxNbSphereByBrick )
	//{
	//	minLevelOfResolutionToHandle++;

	//	minNbSphereByBrick /= powf( nbChildren, minLevelOfResolutionToHandle );
	//}

	//return minLevelOfResolutionToHandle;

	const unsigned int nbChildren = 8;	// for octree
	const unsigned int nbSpheres = static_cast< unsigned int >( static_cast< float >( _nbSpheres ) * powf( nbChildren, _userDefinedMinLevelOfResolutionToHandle ) );
	
	return nbSpheres;
}

///******************************************************************************
// * ...
// ******************************************************************************/
//void SampleCore::setAutomaticMinLevelOfResolutionToHandle( unsigned int pValue )
//{
//	_automaticMinLevelOfResolutionToHandle = pValue;
//
//	// Update DEVICE memory
//	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cAutomaticMinLevelOfResolutionToHandle), &_automaticMinLevelOfResolutionToHandle, sizeof( _automaticMinLevelOfResolutionToHandle ), 0, hipMemcpyHostToDevice ) );
//
//	// Clear the cache
//	clearCache();
//}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getSphereBrickIntersectionType() const
{
	return _sphereBrickIntersectionType;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setSphereBrickIntersectionType( unsigned int pValue )
{
	_sphereBrickIntersectionType = pValue;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cSphereBrickIntersectionType), &_sphereBrickIntersectionType, sizeof( _sphereBrickIntersectionType ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getSphereRadiusFader() const
{
    return _sphereRadiusFader;//_producer->getSphereRadiusFader();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setSphereRadiusFader( float pValue )
{

    //_producer->setSphereRadiusFader( pValue );
    _sphereRadiusFader = pValue;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cSphereRadiusFader), &_sphereRadiusFader, sizeof( _sphereRadiusFader ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasGeometricCriteria() const
{
	return _geometricCriteria;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setGeometricCriteria( bool pFlag )
{
	_geometricCriteria = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cGeometricCriteria), &_geometricCriteria, sizeof( _geometricCriteria ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getMinNbSpheresPerBrick() const
{
	return _minNbSpheresPerBrick;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setMinNbSpheresPerBrick( unsigned int pValue )
{
	_minNbSpheresPerBrick = pValue;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMinNbSpheresPerBrick), &_minNbSpheresPerBrick, sizeof( _minNbSpheresPerBrick ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasScreenBasedCriteria() const
{
	return _screenBasedCriteria;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setScreenBasedCriteria( bool pFlag )
{
	_screenBasedCriteria = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenBasedCriteria), &_screenBasedCriteria, sizeof( _screenBasedCriteria ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasAbsoluteSizeCriteria() const
{
	return _absoluteSizeCriteria;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setAbsoluteSizeCriteria( bool pFlag )
{
	_absoluteSizeCriteria = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cAbsoluteSizeCriteria), &_absoluteSizeCriteria, sizeof( _absoluteSizeCriteria ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasFixedSizeSphere() const
{
	return _fixedSizeSphere;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setFixedSizeSphere( bool pFlag )
{
    _fixedSizeSphere = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cFixedSizeSphere), &_fixedSizeSphere, sizeof( _fixedSizeSphere ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getFixedSizeSphereRadius() const
{
    return _producer->getFixedSizeSphereRadius();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getSphereDiameterCoeff() const
{
    return _sphereDiameterCoeff;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setSphereDiameterCoeff( double pValue )
{
    _sphereDiameterCoeff = pValue;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cCoeffAbsoluteSizeCriteria), &_sphereDiameterCoeff, sizeof( _sphereDiameterCoeff ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getScreenSpaceCoeff() const
{
    return _screenSpaceCoeff;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setScreenSpaceCoeff( unsigned int pValue )
{
    _screenSpaceCoeff = pValue;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenSpaceCoeff), &_screenSpaceCoeff, sizeof( _screenSpaceCoeff ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setFixedSizeSphereRadius( float pValue )
{
	_producer->setFixedSizeSphereRadius( pValue );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasMeanSizeOfSpheres() const
{
	return _meanSizeOfSpheres;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setMeanSizeOfSpheres( bool pFlag )
{
	_meanSizeOfSpheres = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMeanSizeOfSpheres), &_meanSizeOfSpheres, sizeof( _meanSizeOfSpheres ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...alpha
 ******************************************************************************/
bool SampleCore::hasShaderUniformColor() const
{
	return _shaderUseUniformColor;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderUniformColorMode( bool pFlag )
{
	_shaderUseUniformColor = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderUseUniformColor), &_shaderUseUniformColor, sizeof( _shaderUseUniformColor ), 0, hipMemcpyHostToDevice ) );

	// Clear the cache
	clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
const float4& SampleCore::getShaderUniformColor() const
{
	return _shaderUniformColor;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderUniformColor( float pR, float pG, float pB, float pA )
{
	_shaderUniformColor = make_float4( pR, pG, pB, pA );

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderUniformColor), &_shaderUniformColor, sizeof( _shaderUniformColor ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasShaderAnimation() const
{
	return _shaderAnimation;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderAnimation( bool pFlag )
{
	_shaderAnimation = pFlag;

	// Update DEVICE memory with "voxel scale"
	GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderAnimation), &_shaderAnimation, sizeof( _shaderAnimation ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasShaderBlurSphere() const
{
    return _shaderBlurSphere;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderBlurSphere( bool pFlag )
{
    _shaderBlurSphere = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderBlurSphere), &_shaderBlurSphere, sizeof( _shaderBlurSphere ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasShaderFog() const
{
    return _shaderFog;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderFog( bool pFlag )
{
    _shaderFog = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderFog), &_shaderFog, sizeof( _shaderFog ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getFogDensity() const
{

    return _shaderFogDensity;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setFogDensity( float pValue )
{
    _shaderFogDensity = pValue;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderFogDensity), &_shaderFogDensity, sizeof( _shaderFogDensity ), 0, hipMemcpyHostToDevice ) );

}

/******************************************************************************
 * ...
 ******************************************************************************/
const float4& SampleCore::getShaderFogColor() const
{
    return _shaderFogColor;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderFogColor( float pR, float pG, float pB, float pA )
{
    _shaderFogColor = make_float4( pR, pG, pB, pA );

    setClearColor(pR*255, pG*255, pB*255, pA*255);
    //_renderer->setClearColor(make_uchar4(pR*255, pG*255, pB*255, pA*255));

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderFogColor), &_shaderFogColor, sizeof( _shaderFogColor ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::IsLightSourceType() const
{
    return _shaderLightSourceType;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setLightSourceType( bool pFlag )
{
    _shaderLightSourceType = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderLightSourceType), &_shaderLightSourceType, sizeof( _shaderLightSourceType ), 0, hipMemcpyHostToDevice ) );

}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasShading() const
{
    return _shaderFog;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShading( bool pFlag )
{
    _shading = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShading), &_shading, sizeof( _shading ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasBugCorrection() const
{
    return _bugCorrection;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBugCorrection( bool pFlag )
{
    _bugCorrection = pFlag;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cShaderBugCorrection), &_bugCorrection, sizeof( _bugCorrection ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getIlluminationCoeff() const
{
    return _illuminationCoeff;//_producer->getSphereRadiusFader();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setIlluminationCoeff( float pValue )
{
    _illuminationCoeff = pValue;

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cSphereIlluminationCoeff), &_illuminationCoeff, sizeof( _illuminationCoeff ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    //clearCache(); => no need to clear cache ?
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setNumberOfReflections( int pValue ) 
{
	_numberOfReflections = static_cast< unsigned int >( pValue );

    // Update DEVICE memory with "voxel scale"
    GS_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNbMirrorReflections), &_numberOfReflections, sizeof( _numberOfReflections ), 0, hipMemcpyHostToDevice ) );

    // Clear the cache
    //clearCache();
}
