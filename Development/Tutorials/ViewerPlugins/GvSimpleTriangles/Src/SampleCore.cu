#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvPerfMon/GvPerformanceMonitor.h>
#include <GvCore/GvError.h>
#include <GvUtils/GvSimplePriorityPoliciesManagerKernel.h>

// Cuda SDK
#include <hip/hip_vector_types.h>

// Simple Triangles
#include "SampleCore.h"
#include "BvhTree.h"
#include "BvhTreeCache.h"
#include "BvhTreeRenderer.h"
#include "GPUTriangleProducerBVH.h"


// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>
#include <QFileInfo>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;

// GigaVoxels viewer
using namespace GvViewerCore;

// STL
using namespace std;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	GvvPipelineInterface()
,	mColorTex( 0 )
,	mDepthTex( 0 )
,	mFrameBuffer( 0 )
,	mColorBuffer( 0 )
,	mDepthBuffer( 0 )
,	mColorResource( 0 )
,	mDepthResource( 0 )
,	mDisplayOctree( false )
,	mDisplayPerfmon( 0 )
,	mMaxVolTreeDepth( 5 )
,	_filename()
{
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete mBvhTreeRenderer;
	delete mBvhTreeCache;
	delete mBvhTree;
	delete mProducer;

	// Delete the GigaVoxels pipeline
	//delete _pipeline;

	// CUDA tip: clean up to ensure correct profiling
	hipError_t error = hipDeviceReset();
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "SimpleTriangles";
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	if ( ! GvViewerGui::GvvApplication::get().isGPUComputingInitialized() )
	{
		//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
		//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
		hipSetDevice( gpuGetMaxGflopsDeviceId() );
		GV_CHECK_CUDA_ERROR( "hipSetDevice" );
		
		GvViewerGui::GvvApplication::get().setGPUComputingInitialized( true );
	}

	// FIXME: what is that ?
	uint3 volTreePoolRes = make_uint3( BVH_NODE_POOL_SIZE, 1, 1 );
	uint3 vertexPoolRes = make_uint3( BVH_VERTEX_POOL_SIZE, 1, 1 );
	uint3 nodeTileRes = make_uint3( 2, 1, 1 );
	uint3 vertexTileRes = make_uint3( BVH_DATA_PAGE_SIZE, 1, 1 );

	// Instanciate our objects
	mBvhTree = new BvhTreeType( BVH_NODE_POOL_SIZE, BVH_VERTEX_POOL_SIZE );

	// Data production manager
	mBvhTreeCache = new BvhTreeCacheType( mBvhTree, volTreePoolRes, nodeTileRes, vertexPoolRes, vertexTileRes );
	
	// Producer
	mProducer = new ProducerType();
	//mProducer->_filename = std::string( "../../media/meshes/sponza.obj" );
	//mProducer->_filename = std::string( "sponza.obj" );	// ajouter un test sur le fichier d'entr�e (warning, error, etc..)
	//mProducer->_filename = std::string( "J:\\Projects\\Inria\\GigaVoxelsTrunk\\Release\\Bin\\Data\\3DModels\\dabrovic-sponza\\sponza.obj" );	// ajouter un test sur le fichier d'entr�e (warning, error, etc..)
	mProducer->_filename = std::string( "J:\\Projects\\Inria\\GigaVoxelsTrunk\\Release\\Bin\\Data\\3DModels\\stanford_dragon\\dragon.obj" );
	mProducer->initialize( mBvhTree, mBvhTreeCache );
	mBvhTreeCache->addProducer( mProducer );
	
	// Renderer
	mBvhTreeRenderer = new RendererType( mBvhTree, mBvhTreeCache, mProducer );

	// Configure the renderer
	mBvhTreeRenderer->setMaxVolTreeDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT(frame);
	CUDAPM_START_EVENT(app_init_frame);

	glBindFramebuffer(GL_FRAMEBUFFER, mFrameBuffer);

	glClearColor(0.0f, 0.1f, 0.3f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT);

	glEnable(GL_DEPTH_TEST);

	// draw the octree where the sphere will be
	if (mDisplayOctree)
	{
		glPushMatrix();
		glTranslatef(-0.5f, -0.5f, -0.5f);
		//mBvhTreeRenderer->renderFullGL();
		//mProducer->renderFullGL();
		//mProducer->renderGL();
		mProducer->renderDebugGL();
		glPopMatrix();
	}
	
	// copy the current scene into PBO
	glBindBuffer(GL_PIXEL_PACK_BUFFER, mColorBuffer);
	glReadPixels(0, 0, mWidth, mHeight, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glBindBuffer(GL_PIXEL_PACK_BUFFER, mDepthBuffer);
	glReadPixels(0, 0, mWidth, mHeight, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv(GL_MODELVIEW_MATRIX, viewMatrix._array);
	glGetFloatv(GL_PROJECTION_MATRIX, projectionMatrix._array);

	// extract viewport
	GLint params[4];
	glGetIntegerv(GL_VIEWPORT, params);
	int4 viewport = make_int4(params[0], params[1], params[2], params[3]);

	// render the scene into textures
	CUDAPM_STOP_EVENT(app_init_frame);

	// build the world transformation matrix
	float4x4 modelMatrix;

	glPushMatrix();
	glLoadIdentity();
	//glTranslatef(-0.5f, -0.5f, -0.5f);
	glGetFloatv(GL_MODELVIEW_MATRIX, modelMatrix._array);
	glPopMatrix();

	// render
	mBvhTreeRenderer->renderImpl(modelMatrix, viewMatrix, projectionMatrix, viewport);

	// upload changes into the textures
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, mColorBuffer);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mColorTex);
	glTexSubImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, 0, 0, mWidth, mHeight, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	/*glBindBuffer(GL_PIXEL_UNPACK_BUFFER, mDepthBuffer);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mDepthTex);
	glTexSubImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, 0, 0, mWidth, mHeight, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);*/

	// render the result to the screen
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();

	glEnable(GL_TEXTURE_RECTANGLE_EXT);
	glDisable(GL_DEPTH_TEST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mColorTex);

	GLint sMin = 0;
	GLint tMin = 0;
	GLint sMax = mWidth;
	GLint tMax = mHeight;

	glBegin(GL_QUADS);
		glColor3f(1.0f, 1.0f, 1.0f);
		glTexCoord2i(sMin, tMin); glVertex2i(-1, -1);
		glTexCoord2i(sMax, tMin); glVertex2i( 1, -1);
		glTexCoord2i(sMax, tMax); glVertex2i( 1,  1);
		glTexCoord2i(sMin, tMax); glVertex2i(-1,  1);
	glEnd();

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);

	glDisable(GL_TEXTURE_RECTANGLE_EXT);

	glPopMatrix();
	glMatrixMode(GL_MODELVIEW);
	glPopMatrix();

	mBvhTreeRenderer->nextFrame();

	CUDAPM_STOP_EVENT(frame);
	CUDAPM_STOP_FRAME;

	if (mDisplayPerfmon)
		GvPerfMon::CUDAPerfMon::get().displayFrameGL(mDisplayPerfmon - 1);
}

/******************************************************************************
 * ...
 *
 * @param width ...
 * @param height ...
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	mWidth = width;
	mHeight = height;

	// Re-init Perfmon subsystem
	CUDAPM_RESIZE(make_uint2(mWidth, mHeight));

	// Create frame-dependent objects
	if (mColorResource)
		GV_CUDA_SAFE_CALL(hipGraphicsUnregisterResource(mColorResource));
	if (mDepthResource)
		GV_CUDA_SAFE_CALL(hipGraphicsUnregisterResource(mDepthResource));

	if (mColorBuffer)
		glDeleteBuffers(1, &mColorBuffer);
	if (mDepthBuffer)
		glDeleteBuffers(1, &mDepthBuffer);

	if (mColorTex)
		glDeleteTextures(1, &mColorTex);
	if (mDepthTex)
		glDeleteTextures(1, &mDepthTex);

	if (mFrameBuffer)
		glDeleteFramebuffers(1, &mFrameBuffer);

	glGenBuffers(1, &mColorBuffer);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, mColorBuffer);
	glBufferData(GL_PIXEL_PACK_BUFFER, width * height * sizeof(GLubyte) * 4, NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glGenTextures(1, &mColorTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mColorTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenBuffers(1, &mDepthBuffer);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, mDepthBuffer);
	glBufferData(GL_PIXEL_PACK_BUFFER, width * height * sizeof(GLuint), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glGenTextures(1, &mDepthTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, mDepthTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_DEPTH24_STENCIL8_EXT, width, height, 0, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenFramebuffers(1, &mFrameBuffer);
	glBindFramebuffer(GL_FRAMEBUFFER, mFrameBuffer);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_RECTANGLE_EXT, mColorTex, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, mDepthTex, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_STENCIL_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, mDepthTex, 0);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	GV_CHECK_GL_ERROR();

	// Create CUDA resources from OpenGL objects
	GV_CUDA_SAFE_CALL(hipGraphicsGLRegisterBuffer(&mColorResource, mColorBuffer, hipGraphicsRegisterFlagsNone));
	GV_CUDA_SAFE_CALL(hipGraphicsGLRegisterBuffer(&mDepthResource, mDepthBuffer, hipGraphicsRegisterFlagsNone));

	//// Pass resources to the renderer
	mBvhTreeRenderer->setColorResource(mColorResource);
	mBvhTreeRenderer->setDepthResource(mDepthResource);
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::clearCache()
{
	//mBvhTreeRenderer->clearCache();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	mDisplayOctree = !mDisplayOctree;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	//mBvhTreeRenderer->dynamicUpdateState() = !mBvhTreeRenderer->dynamicUpdateState();
}

/******************************************************************************
 * ...
 *
 * @param mode ...
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if (mDisplayPerfmon)
		mDisplayPerfmon = 0;
	else
		mDisplayPerfmon = mode;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if (mMaxVolTreeDepth < 32)
		mMaxVolTreeDepth++;

	//mBvhTreeRenderer->setMaxVolTreeDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if (mMaxVolTreeDepth > 0)
		mMaxVolTreeDepth--;

	//mBvhTreeRenderer->setMaxVolTreeDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::has3DModel() const
{
	return true;
}

/******************************************************************************
 * Get the 3D model filename to load
 *
 * @return the 3D model filename to load
 ******************************************************************************/
string SampleCore::get3DModelFilename() const
{
	return _filename;
}

/******************************************************************************
 * Set the 3D model filename to load
 *
 * @param pFilename the 3D model filename to load
 ******************************************************************************/
void SampleCore::set3DModelFilename( const string& pFilename )
{
	_filename = pFilename;

	// ---- Delete the 3D scene if needed ----
	
	//if ( _proxyGeometry != NULL )
	//{
	//	delete _proxyGeometry;
	//	_proxyGeometry = NULL;

	//	// Clear the GigaVoxels cache
	//	_pipeline->editCache()->clearCache();
	//}

	//// Initialize proxy geometry (load the 3D scene)
	////
	//// - find a way to modify internal buffer size
	//_proxyGeometry = new ProxyGeometry();
	//_proxyGeometry->set3DModelFilename( pFilename );
	//_proxyGeometry->initialize();
	//// Restore previous proxy geometry state
	//_proxyGeometry->setScreenBasedCriteria( screenBasedCriteria );
	//_proxyGeometry->setScreenBasedCriteriaCoefficient( screenBasedCriteriaCoefficient );
	//_proxyGeometry->setMaterialAlphaCorrectionCoefficient( materialAlphaCorrectionCoefficient );
	//_pipeline->editRenderer()->setProxyGeometry( _proxyGeometry );
	//// Reset proxy geometry resources
	//_pipeline->editRenderer()->unregisterProxyGeometryGraphicsResources();
	//_proxyGeometry->setBufferSize( _width, _height );
	//_pipeline->editRenderer()->registerProxyGeometryGraphicsResources();
	//// Noise parameters
	//setNoiseFirstFrequency( _noiseFirstFrequency );
	//setNoiseStrength( _noiseStrength );
}
