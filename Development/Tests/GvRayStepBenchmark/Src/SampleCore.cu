#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvStructure/GvDataProductionManager.h>
#include <GvRendering/GvGraphicsInteroperabiltyHandler.h>
#include <GvUtils/GvSimplePipeline.h>
#include <GvUtils/GvSimpleHostProducer.h>
#include <GvUtils/GvSimpleHostShader.h>
#include <GvUtils/GvCommonGraphicsPass.h>
#include <GvCore/GvError.h>
#include <GvPerfMon/GvPerformanceMonitor.h>
#include <GvUtils/GvShaderProgram.h>

// Project
#include "ProducerKernel.h"
#include "ShaderKernel.h"
#include "VolumeTreeRendererCUDA.h"

// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>

// Cuda SDK
#include <hip/hip_vector_types.h>

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>
#include <QFileInfo>
#include <QImage>
#include <QGLWidget>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;

// GigaVoxels viewer
using namespace GvViewerCore;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	GvvPipelineInterface()
,	_pipeline( NULL )
,	_graphicsEnvironment( NULL )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 0 )
,	_depthBuffer( 0 )
,	_colorTex( 0 )
,	_colorRenderBuffer( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_shaderProgram( NULL )
,	_positionBuffer( 0 )
,	_vao( 0 )
{
	// Translation used to position the GigaVoxels data structure
	_translation[ 0 ] = -0.5f;
	_translation[ 1 ] = -0.5f;
	_translation[ 2 ] = -0.5f;

	// Light position
	_lightPosition = make_float3( 1.f, 1.f, 1.f );

	// Renderer's parameters
	_hasRendererParametersActivated = false;
	_rendererRayStep = 0.0f;
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	// Delete shader program
	delete _shaderProgram;

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();

	// Delete the GigaVoxels pipeline
	delete _pipeline;

	// CUDA tip: clean up to ensure correct profiling
	//hipError_t error = hipDeviceReset();
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "SimpleSphere";
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	if ( ! GvViewerGui::GvvApplication::get().isGPUComputingInitialized() )
	{
		//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
		//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
		hipSetDevice( gpuGetMaxGflopsDeviceId() );
		GV_CHECK_CUDA_ERROR( "hipSetDevice" );
		
		GvViewerGui::GvvApplication::get().setGPUComputingInitialized( true );
	}

	// Pipeline creation
	_pipeline = new PipelineType();
	ProducerType* producer = new ProducerType();
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, producer, shader );

	// Pipeline configuration
	_maxVolTreeDepth = 5;
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );

	// Graphics environment creation
	_graphicsEnvironment = new GvCommonGraphicsPass();

	// Custom initialization
	// Note : this could be done via an XML settings file loaded at initialization
	setRendererParametersActivated( false );
	setRendererRayStep( 0.01f );

	// Shaders data repository
	QString dataRepository = QCoreApplication::applicationDirPath();
	dataRepository += QDir::separator();
	dataRepository += QString( "Data" );
	dataRepository += QDir::separator();
	dataRepository += QString( "Shaders" );
	dataRepository += QDir::separator();
	dataRepository += QString( "GvSimpleSphere" );
	dataRepository += QDir::separator();
	QString vertexShaderFilename = dataRepository + QString( "vertex.glsl" );
	QString fragmentShaderFilename = dataRepository + QString( "fragment.glsl" );

	// Initialize shader program
	_shaderProgram = new GvShaderProgram();
	_shaderProgram->addShader( GvShaderProgram::eVertexShader, vertexShaderFilename.toStdString() );
	_shaderProgram->addShader( GvShaderProgram::eFragmentShader, fragmentShaderFilename.toStdString() );
	_shaderProgram->link();

	//@todo use a user customizable constant 
	const unsigned int nbVertices = 4;

	// Vertex position buffer initialization
	float positions[] =
	{
		-0.5f, -0.5f, 0.0f,
		0.5f, -0.5f, 0.0f,
		-0.5f, 0.5f, 0.0f,
		0.5f, 0.5f, 0.0f
	};
	glGenBuffers( 1, &_positionBuffer );
	glBindBuffer( GL_ARRAY_BUFFER, _positionBuffer );
	GLsizeiptr vertexBufferSize = sizeof( GLfloat ) * nbVertices * 3;
	//glBufferData( GL_ARRAY_BUFFER, vertexBufferSize, NULL, GL_DYNAMIC_DRAW );
	glBufferData( GL_ARRAY_BUFFER, vertexBufferSize, positions, GL_STATIC_DRAW );
	glBindBuffer( GL_ARRAY_BUFFER, 0 );

	// Vertex array object initialization
	glGenVertexArrays( 1, &_vao );
	glBindVertexArray( _vao );
	glEnableVertexAttribArray( 0 );	// vertex position
	glBindBuffer( GL_ARRAY_BUFFER, _positionBuffer );
	glVertexAttribPointer( 0/*attribute index*/, 3/*nb components per vertex*/, GL_FLOAT/*type*/, GL_FALSE/*un-normalized*/, 0/*memory stride*/, static_cast< GLubyte* >( NULL )/*byte offset from buffer*/ );
	glBindBuffer( GL_ARRAY_BUFFER, 0 );
	glBindVertexArray( 0 );
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	glMatrixMode( GL_MODELVIEW );

	// Handle image downscaling if activated
	int bufferWidth = _graphicsEnvironment->getBufferWidth();
	int bufferHeight = _graphicsEnvironment->getBufferHeight();
	if ( hasImageDownscaling() )
	{
		bufferWidth = _graphicsEnvironment->getImageDownscalingWidth();
		bufferHeight = _graphicsEnvironment->getImageDownscalingHeight();
		glViewport( 0, 0, bufferWidth, bufferHeight );
	}

	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		// Translation used to position the GigaVoxels data structure
		glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
		_pipeline->editDataStructure()->displayDebugOctree();
		glPopMatrix();
		{
			//// Display VBO
			//_shaderProgram->use();
			//glEnable( GL_PROGRAM_POINT_SIZE );
			//glPointSize( 15.0f );
			//// Extract view transformations
			//float4x4 modelViewMatrix;
			//float4x4 projectionMatrix;
			//glGetFloatv( GL_MODELVIEW_MATRIX, modelViewMatrix._array );
			//glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );
			//GLuint location = glGetUniformLocation( _shaderProgram->_program, "uModelViewMatrix" );
			//if ( location >= 0 )
			//{
			//	glUniformMatrix4fv( location, 1, GL_FALSE, modelViewMatrix._array );
			//}
			//location = glGetUniformLocation( _shaderProgram->_program, "uProjectionMatrix" );
			//if ( location >= 0 )
			//{
			//	glUniformMatrix4fv( location, 1, GL_FALSE, projectionMatrix._array );
			//}
			//glBindVertexArray( _vao );
			////glDrawArrays( GL_POINTS, 0, 4 );
			//glDrawArrays( GL_TRIANGLE_STRIP, 0, 4 );
			//glBindVertexArray( 0 );
			//glDisable( GL_PROGRAM_POINT_SIZE );
			//glUseProgram( 0 );
		}
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
		glReadPixels( 0, 0, bufferWidth, bufferHeight, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		glClear( GL_COLOR_BUFFER_BIT );
	}
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// Extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

	// Extract viewport
	GLint params[ 4 ];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4( params[ 0 ], params[ 1 ], params[ 2 ], params[ 3 ] );
	// Handle image downscaling if activated
	if ( hasImageDownscaling() )
	{
		// TO DO : clean this... it would better to send real viewport info and retrieve realBufferSize in the renderer ?
		viewport.z = bufferWidth;
		viewport.w = bufferHeight;
	}

	// render the scene into textures
	CUDAPM_STOP_EVENT( app_init_frame );

	// Build the world transformation matrix
	float4x4 modelMatrix;
	glPushMatrix();
	glLoadIdentity();
	// Translation used to position the GigaVoxels data structure
	glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// Render
	_pipeline->editRenderer()->render( modelMatrix, viewMatrix, projectionMatrix, viewport );
	
	if ( _graphicsEnvironment->getType() != 0 )
	{
		// Copy a block of pixels from the read framebuffer to the draw framebuffer
		//glBindFramebuffer( GL_DRAW_FRAMEBUFFER, 0 );	// already done before
		glBindFramebuffer( GL_READ_FRAMEBUFFER, _frameBuffer );
		//glReadBuffer( GL_COLOR_ATTACHMENT0 + TextureType ); => not use because we only have one color attachment
		/*GLint srcX0 = 0;
		GLint srcY0 = 0;
		GLint srcX1 = bufferWidth;
		GLint srcY1 = bufferHeight;
		GLint dstX0 = 0;
		GLint dstY0 = 0;
		GLint dstX1 = bufferWidth;
		GLint dstY1 = bufferHeight;
		GLbitfield mask = GL_COLOR_BUFFER_BIT;
		GLenum filter = GL_NEAREST;*/
		// Handle image downscaling if activated
		if ( hasImageDownscaling() )
		{
			//glBlitFramebuffer( srcX0, srcY0, srcX1, srcY1, dstX0, dstY0, dstX1, dstY1, mask, filter );

			int bufferWidth = _graphicsEnvironment->getBufferWidth();
			int bufferHeight = _graphicsEnvironment->getBufferHeight();
			int imageDownscalingWidth = _graphicsEnvironment->getImageDownscalingWidth();
			int imageDownscalingHeight = _graphicsEnvironment->getImageDownscalingHeight();
			glViewport( 0, 0, bufferWidth, bufferHeight );
			glBlitFramebuffer( 0, 0, imageDownscalingWidth, imageDownscalingHeight, 0, 0, bufferWidth, bufferHeight, GL_COLOR_BUFFER_BIT, GL_LINEAR );
		}
		else
		{
			//glBlitFramebuffer( srcX0, srcY0, srcX1, srcY1, dstX0, dstY0, dstX1, dstY1, mask, filter );
			
			glBlitFramebuffer( 0, 0, bufferWidth, bufferHeight, 0, 0, bufferWidth, bufferHeight, GL_COLOR_BUFFER_BIT, GL_NEAREST );
		}
	}
	else
	{
		// Render the result to the screen
		glMatrixMode( GL_MODELVIEW );
		glPushMatrix();
		glLoadIdentity();

		glMatrixMode( GL_PROJECTION );
		glPushMatrix();
		glLoadIdentity();

		glEnable( GL_TEXTURE_RECTANGLE_EXT );
		glDisable( GL_DEPTH_TEST );

		glActiveTexture( GL_TEXTURE0 );
		glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );
	
		// Handle image downscaling if activated
		if ( hasImageDownscaling() )
		{
			glViewport( 0, 0, _graphicsEnvironment->getBufferWidth(), _graphicsEnvironment->getBufferHeight() );
		}

		GLint sMin = 0;
		GLint tMin = 0;
		GLint sMax = bufferWidth;
		GLint tMax = bufferHeight;

		glBegin( GL_QUADS );
			glColor3f( 1.0f, 1.0f, 1.0f );
			glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
			glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
			glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
			glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
		glEnd();

		glActiveTexture( GL_TEXTURE0 );
		glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );

		glDisable( GL_TEXTURE_RECTANGLE_EXT );

		glPopMatrix();
		glMatrixMode( GL_MODELVIEW );
		glPopMatrix();
	}

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	//_pipeline->editRenderer()->doPostRender();
	
	// Update GigaVoxels info
	_pipeline->editRenderer()->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	if ( _displayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::getApplicationPerfMon().displayFrameGL( _displayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param pWidth the new width
 * @param pHeight the new height
 ******************************************************************************/
void SampleCore::resize( int pWidth, int pHeight )
{
	// LOG
	//
	// @todo : check and avoid 0 values, replace by 1 and warn user
	if ( pWidth == 0 )
	{
		// TO DO
		// ...
	}
	if ( pHeight == 0 )
	{
		// TO DO
		// ...
	}

	// --------------------------
	// Reset default active frame region for rendering
	_pipeline->editRenderer()->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );
	// --------------------------

	// Update graphics environment
	_graphicsEnvironment->setBufferSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->editRenderer()->clearCache();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();

	if ( _displayOctree )
	{
		if ( _graphicsEnvironment->getType() == 0 )
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		}
		else
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorRenderBuffer, GL_RENDERBUFFER );
		}
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		if ( _graphicsEnvironment->getType() == 0 )
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		}
		else
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorRenderBuffer, GL_RENDERBUFFER );
		}
	}
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	setDynamicUpdate( ! hasDynamicUpdate() );
}

/******************************************************************************
 * Get the dynamic update state
 *
 * @return the dynamic update state
 ******************************************************************************/
bool SampleCore::hasDynamicUpdate() const
{
	return _pipeline->editRenderer()->dynamicUpdateState();
}

/******************************************************************************
 * Set the dynamic update state
 *
 * @param pFlag the dynamic update state
 ******************************************************************************/
void SampleCore::setDynamicUpdate( bool pFlag )
{
	_pipeline->editRenderer()->dynamicUpdateState() = pFlag;

	//-------------------------------
	// TEST
	unsigned int type = ( _graphicsEnvironment->getType() + 1 ) % 2;
	std::cout << "type = " << type << std::endl;
	_graphicsEnvironment->setType( type );
	resetGraphicsresources();
	//-------------------------------
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Get the max depth.
 *
 * @return the max depth
 ******************************************************************************/
unsigned int SampleCore::getRendererMaxDepth() const
{
	return _pipeline->editDataStructure()->getMaxDepth();
}

/******************************************************************************
 * Set the max depth.
 *
 * @param pValue the max depth
 ******************************************************************************/
void SampleCore::setRendererMaxDepth( unsigned int pValue )
{
	_pipeline->editDataStructure()->setMaxDepth( pValue );
}

/******************************************************************************
 * Set the request strategy indicating if, during data structure traversal,
 * priority of requests is set on brick loads or on node subdivisions first.
 *
 * @param pFlag the flag indicating the request strategy
 ******************************************************************************/
void SampleCore::setRendererPriorityOnBricks( bool pFlag )
{
	_pipeline->editRenderer()->setPriorityOnBricks( pFlag );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	_pipeline->editRenderer()->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::hasLight() const
{
	return true;
}

/******************************************************************************
 * Get the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::getLightPosition( float& pX, float& pY, float& pZ ) const
{
	pX = _lightPosition.x;
	pY = _lightPosition.y;
	pZ = _lightPosition.z;
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	// WARNING
	// Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
	// Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
	_lightPosition.x = pX/* - _translation[ 0 ]*/;
	_lightPosition.y = pY/* - _translation[ 1 ]*/;
	_lightPosition.z = pZ/* - _translation[ 2 ]*/;

	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &_lightPosition, sizeof( _lightPosition ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the translation used to position the GigaVoxels data structure
 *
 * @param pX the x componenet of the translation
 * @param pX the y componenet of the translation
 * @param pX the z componenet of the translation
 ******************************************************************************/
void SampleCore::getTranslation( float& pX, float& pY, float& pZ ) const
{
	pX = _translation[ 0 ];
	pY = _translation[ 1 ];
	pZ = _translation[ 2 ];
}

/******************************************************************************
 * Get the node cache memory
 *
 * @return the node cache memory
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheMemory() const
{
	return NODEPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the node cache memory
 *
 * @param pValue the node cache memory
 ******************************************************************************/
void SampleCore::setNodeCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache memory
 *
 * @return the brick cache memory
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheMemory() const
{
	return BRICKPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the brick cache memory
 *
 * @param pValue the brick cache memory
 ******************************************************************************/
void SampleCore::setBrickCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the node cache capacity
 *
 * @return the node cache capacity
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheCapacity() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the node cache capacity
 *
 * @param pValue the node cache capacity
 ******************************************************************************/
void SampleCore::setNodeCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache capacity
 *
 * @return the brick cache capacity
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheCapacity() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the brick cache capacity
 *
 * @param pValue the brick cache capacity
 ******************************************************************************/
void SampleCore::setBrickCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the number of unused nodes in cache
 *
 * @return the number of unused nodes in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedNodes() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Get the number of unused bricks in cache
 *
 * @return the number of unused bricks in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedBricks() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Get the node tile resolution of the data structure.
 *
 * @param pX the X node tile resolution
 * @param pY the Y node tile resolution
 * @param pZ the Z node tile resolution
 ******************************************************************************/
void SampleCore::getDataStructureNodeTileResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& nodeTileResolution = _pipeline->getDataStructure()->getNodeTileResolution().get();

	pX = nodeTileResolution.x;
	pY = nodeTileResolution.y;
	pZ = nodeTileResolution.z;
}

/******************************************************************************
 * Get the brick resolution of the data structure (voxels).
 *
 * @param pX the X brick resolution
 * @param pY the Y brick resolution
 * @param pZ the Z brick resolution
 ******************************************************************************/
void SampleCore::getDataStructureBrickResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& brickResolution = _pipeline->getDataStructure()->getBrickResolution().get();

	pX = brickResolution.x;
	pY = brickResolution.y;
	pZ = brickResolution.z;
}

/******************************************************************************
 * Get the max number of requests of node subdivisions.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbNodeSubdivisions() const
{
	return _pipeline->getCache()->getMaxNbNodeSubdivisions();
}

/******************************************************************************
 * Set the max number of requests of node subdivisions.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbNodeSubdivisions( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbNodeSubdivisions( pValue );
}

/******************************************************************************
 * Get the max number of requests of brick of voxel loads.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbBrickLoads() const
{
	return _pipeline->getCache()->getMaxNbBrickLoads();
}

/******************************************************************************
 * Set the max number of requests of brick of voxel loads.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbBrickLoads( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbBrickLoads( pValue );
}

/******************************************************************************
 * Get the number of requests of node subdivisions the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbNodeSubdivisionRequests() const
{
	return _pipeline->getCache()->getNbNodeSubdivisionRequests();
}

/******************************************************************************
 * Get the number of requests of brick of voxel loads the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbBrickLoadRequests() const
{
	return _pipeline->getCache()->getNbBrickLoadRequests();
}

/******************************************************************************
 * Get the cache policy
 *
 * @return the cache policy
 ******************************************************************************/
unsigned int SampleCore::getCachePolicy() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getPolicy();
}

/******************************************************************************
 * Set the cache policy
 *
 * @param pValue the cache policy
 ******************************************************************************/
void SampleCore::setCachePolicy( unsigned int pValue )
{
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( static_cast< PipelineType::CacheType::NodesCacheManager::ECachePolicy>( pValue ) );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( static_cast< PipelineType::CacheType::BricksCacheManager::ECachePolicy>( pValue ) );
}

/******************************************************************************
 * Get the nodes cache usage
 *
 * @return the nodes cache usage
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheUsage() const
{
	//const unsigned int nbProducedElements = _pipeline->getCache()->getNodesCacheManager()->_totalNumLoads;
	const unsigned int nbProducedElements = _pipeline->getCache()->getNodesCacheManager()->_numElemsNotUsed;
	const unsigned int nbElements = _pipeline->getCache()->getNodesCacheManager()->getNumElements();

	const unsigned int cacheUsage = static_cast< unsigned int >( 100.0f * static_cast< float >( nbElements - nbProducedElements ) / static_cast< float >( nbElements ) );

	//std::cout << "NODE cache usage [ " << nbProducedElements << " / "<< nbElements << " : " << cacheUsage << std::endl;

	return cacheUsage;
}

/******************************************************************************
 * Get the bricks cache usage
 *
 * @return the bricks cache usage
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheUsage() const
{
	//const unsigned int nbProducedElements = _pipeline->getCache()->getBricksCacheManager()->_totalNumLoads;
	const unsigned int nbProducedElements = _pipeline->getCache()->getBricksCacheManager()->_numElemsNotUsed;
	const unsigned int nbElements = _pipeline->getCache()->getBricksCacheManager()->getNumElements();

	const unsigned int cacheUsage = static_cast< unsigned int >( 100.0f * static_cast< float >( nbElements - nbProducedElements ) / static_cast< float >( nbElements ) );

	//std::cout << "BRICK cache usage [ " << nbProducedElements << " / "<< nbElements << " : " << cacheUsage << std::endl;

	return cacheUsage;
}

/******************************************************************************
 * Tell wheter or not the pipeline uses image downscaling.
 *
 * @return the flag telling wheter or not the pipeline uses image downscaling
 ******************************************************************************/
bool SampleCore::hasImageDownscaling() const
{
	return _graphicsEnvironment->hasImageDownscaling();
}

/******************************************************************************
 * Set the flag telling wheter or not the pipeline uses image downscaling
 *
 * @param pFlag the flag telling wheter or not the pipeline uses image downscaling
 ******************************************************************************/
void SampleCore::setImageDownscaling( bool pFlag )
{
	// Update graphics environment
	_graphicsEnvironment->setImageDownscaling( pFlag );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Get the internal graphics buffer size
 *
 * @param pWidth the internal graphics buffer width
 * @param pHeight the internal graphics buffer height
 ******************************************************************************/
void SampleCore::getViewportSize( unsigned int& pWidth, unsigned int& pHeight ) const
{
	if ( _graphicsEnvironment != NULL )
	{
		pWidth = static_cast< unsigned int >( _graphicsEnvironment->getBufferWidth() );
		pHeight = static_cast< unsigned int >( _graphicsEnvironment->getBufferHeight() );
	}
}

/******************************************************************************
 * Set the internal graphics buffer size
 *
 * @param pWidth the internal graphics buffer width
 * @param pHeight the internal graphics buffer height
 ******************************************************************************/
void SampleCore::setViewportSize( unsigned int pWidth, unsigned int pHeight )
{
	// --------------------------
	// Reset default active frame region for rendering
	_pipeline->editRenderer()->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );
	// --------------------------

	// Update graphics environment
	_graphicsEnvironment->setBufferSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Get the internal graphics buffer size
 *
 * @param pWidth the internal graphics buffer width
 * @param pHeight the internal graphics buffer height
 ******************************************************************************/
void SampleCore::getGraphicsBufferSize( unsigned int& pWidth, unsigned int& pHeight ) const
{
	if ( _graphicsEnvironment != NULL )
	{
		pWidth = static_cast< unsigned int >( _graphicsEnvironment->getImageDownscalingWidth() );
		pHeight = static_cast< unsigned int >( _graphicsEnvironment->getImageDownscalingHeight() );
	}
}

/******************************************************************************
 * Set the internal graphics buffer size
 *
 * @param pWidth the internal graphics buffer width
 * @param pHeight the internal graphics buffer height
 ******************************************************************************/
void SampleCore::setGraphicsBufferSize( unsigned int pWidth, unsigned int pHeight )
{
	// --------------------------
	// Reset default active frame region for rendering
	_pipeline->editRenderer()->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );
	// --------------------------

	// Update graphics environment
	_graphicsEnvironment->setImageDownscalingSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Reset graphics resources
 ******************************************************************************/
void SampleCore::resetGraphicsresources()
{
	// [ 1 ] - Reset graphics resources

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();
	
	// Update graphics environment
	_graphicsEnvironment->reset();
	
	// Update internal variables
	_depthBuffer = _graphicsEnvironment->getDepthBuffer();
	_colorTex = _graphicsEnvironment->getColorTexture();
	_colorRenderBuffer = _graphicsEnvironment->getColorRenderBuffer();
	_depthTex = _graphicsEnvironment->getDepthTexture();
	_frameBuffer = _graphicsEnvironment->getFrameBuffer();
	
	// [ 2 ] - Connect graphics resources

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		if ( _graphicsEnvironment->getType() == 0 )
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		}
		else
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorRenderBuffer, GL_RENDERBUFFER );
		}
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		if ( _graphicsEnvironment->getType() == 0 )
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		}
		else
		{
			_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorRenderBuffer, GL_RENDERBUFFER );
		}
	}
}

/******************************************************************************
 * Tell wheter or not pipeline uses programmable shaders
 *
 * @return a flag telling wheter or not pipeline uses programmable shaders
 ******************************************************************************/
bool SampleCore::hasProgrammableShaders() const
{
	return true;
}

/******************************************************************************
 * Tell wheter or not pipeline has a given type of shader
 *
 * @param pShaderType the type of shader to test
 *
 * @return a flag telling wheter or not pipeline has a given type of shader
 ******************************************************************************/
bool SampleCore::hasShaderType( unsigned int pShaderType ) const
{
	return _shaderProgram->hasShaderType( static_cast< GvShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * Get the source code associated to a given type of shader
 *
 * @param pShaderType the type of shader
 *
 * @return the associated shader source code
 ******************************************************************************/
std::string SampleCore::getShaderSourceCode( unsigned int pShaderType ) const
{
	return _shaderProgram->getShaderSourceCode( static_cast< GvShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * Get the filename associated to a given type of shader
 *
 * @param pShaderType the type of shader
 *
 * @return the associated shader filename
 ******************************************************************************/
std::string SampleCore::getShaderFilename( unsigned int pShaderType ) const
{
	return _shaderProgram->getShaderFilename( static_cast< GvShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * ...
 *
 * @param pShaderType the type of shader
 *
 * @return ...
 ******************************************************************************/
bool SampleCore::reloadShader( unsigned int pShaderType )
{
	return _shaderProgram->reloadShader( static_cast< GvShaderProgram::ShaderType >( pShaderType ) );
}

/******************************************************************************
 * Tell wheter or not the renderer's custom parameters are activated
 *
 * @return flag to tell wheter or not the renderer's custom parameters are activated
 ******************************************************************************/
bool SampleCore::hasRendererParametersActivated() const
{
	return _hasRendererParametersActivated;
}

/******************************************************************************
 * Set the renderer's custom parameters activated
 *
 * @param pFlag flag to tell wheter or not the renderer's custom parameters are activated
 ******************************************************************************/
void SampleCore::setRendererParametersActivated( bool pFlag )
{
	_hasRendererParametersActivated = pFlag;
	
	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cHasRendererParametersActivated), &_hasRendererParametersActivated, sizeof( _hasRendererParametersActivated ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the renderer's ray step
 *
 * @return the renderer's ray step
 ******************************************************************************/
float SampleCore::getRendererRayStep() const
{
	return _rendererRayStep;
}

/******************************************************************************
 * Set the renderer's ray step
 *
 * @param pValue the renderer's ray step
 ******************************************************************************/
void SampleCore::setRendererRayStep( float pValue )
{
	_rendererRayStep = pValue;
	
	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cRendererRayStep), &_rendererRayStep, sizeof( _rendererRayStep ), 0, hipMemcpyHostToDevice ) );
}
