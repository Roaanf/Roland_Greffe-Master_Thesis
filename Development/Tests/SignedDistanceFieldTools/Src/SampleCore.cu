#include "hip/hip_runtime.h"
/*
 * GigaVoxels - GigaSpace
 *
 * Website: http://gigavoxels.inrialpes.fr/
 *
 * Contributors: GigaVoxels Team
 *
 * Copyright (C) 2007-2015 INRIA - LJK (CNRS - Grenoble University), All rights reserved.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvStructure/GvDataProductionManager.h>
#include <GvRendering/GvRendererCUDA.h>
#include <GvUtils/GvSimplePipeline.h>
#include <GvUtils/GvSimpleHostProducer.h>
#include <GvUtils/GvSimpleHostShader.h>
#include <GvUtils/GvCommonGraphicsPass.h>
#include <GvUtils/GvTransferFunction.h>
#include <GvCore/GvError.h>
#include <GvPerfMon/GvPerformanceMonitor.h>

// Project
#include "ProducerKernel.h"
#include "ShaderKernel.h"
#include "vdCube3D4.h"
#include "sdfReader.h"

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>
#include <QFileInfo>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		//   8 Mo
#define BRICKPOOL_MEMSIZE	( 384U * 1024U * 1024U )	// 384 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	_graphicsEnvironment( NULL )
,	_depthBuffer( 0 )
,	_colorTex( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_width( 512 )
,	_height( 512 )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 6 )
,	_signedDistanceField( NULL )
,	_transferFunction( NULL )
{
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	// Finalize the GigaVoxels pipeline (free memory)
	finalizePipeline();

	// Finalize the 3D model (free memory)
	finalize3DModel();

	// Finalize the transfer function (free memory)
	finalizeTransferFunction();

	// Finalize graphics resources
	finalizeGraphicsResources();
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
	//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
	hipSetDevice( gpuGetMaxGflopsDeviceId() );
	GV_CHECK_CUDA_ERROR( "hipSetDevice" );

	// Initialize the GigaVoxels pipeline
	initializePipeline();

	// Initialize the 3D model
	initialize3DModel();

	// Initialize the transfer function
	initializeTransferFunction();
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );

	glMatrixMode( GL_MODELVIEW );

	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		glTranslatef( -0.5f, -0.5f, -0.5f );
		_pipeline->editDataStructure()->displayDebugOctree();
		glPopMatrix();
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
		glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		glClear( GL_COLOR_BUFFER_BIT );
	}

	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

	// extract viewport
	GLint params[4];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4(params[0], params[1], params[2], params[3]);

	// render the scene into textures
	CUDAPM_STOP_EVENT( app_init_frame );

	// Build the world transformation matrix
	float4x4 modelMatrix;
	glPushMatrix();
	glLoadIdentity();
	glTranslatef( -0.5f, -0.5f, -0.5f );
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// Render
	_pipeline->editRenderer()->render( modelMatrix, viewMatrix, projectionMatrix, viewport );

	// Render the result to the screen
	glMatrixMode( GL_MODELVIEW );
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode( GL_PROJECTION );
	glPushMatrix();
	glLoadIdentity();

	glDisable( GL_DEPTH_TEST );
	glEnable( GL_TEXTURE_RECTANGLE_EXT );
	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );

	// Draw a full screen quad
	GLint sMin = 0;
	GLint tMin = 0;
	GLint sMax = _width;
	GLint tMax = _height;
	glBegin( GL_QUADS );
	glColor3f( 1.0f, 1.0f, 1.0f );
	glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
	glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
	glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
	glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
	glEnd();

	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );
	glDisable( GL_TEXTURE_RECTANGLE_EXT );
	
	glPopMatrix();
	glMatrixMode( GL_MODELVIEW );
	glPopMatrix();

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	//_volumeTreeRenderer->doPostRender();
	
	// Update GigaVoxels info
	_pipeline->editRenderer()->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the GigaVoxels performance monitor (if it has been activated during GigaVoxels compilation)
	if ( _displayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::getApplicationPerfMon().displayFrameGL( _displayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int pWidth, int pHeight )
{
	// LOG
	//
	// @todo : check and avoid 0 values, replace by 1 and warn user
	if ( pWidth == 0 )
	{
		// TO DO
		// ...
	}
	if ( pHeight == 0 )
	{
		// TO DO
		// ...
	}

	_width = pWidth;
	_height = pHeight;

	// --------------------------
	// Reset default active frame region for rendering
	_pipeline->editRenderer()->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );
	// --------------------------

	// Update graphics environment
	_graphicsEnvironment->setBufferSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Reset graphics resources
 ******************************************************************************/
void SampleCore::resetGraphicsresources()
{
	// [ 1 ] - Reset graphics resources

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();
	
	// Update graphics environment
	_graphicsEnvironment->reset();
	
	// Update internal variables
	_depthBuffer = _graphicsEnvironment->getDepthBuffer();
	_colorTex = _graphicsEnvironment->getColorTexture();
	_depthTex = _graphicsEnvironment->getDepthTexture();
	_frameBuffer = _graphicsEnvironment->getFrameBuffer();
	
	// [ 2 ] - Connect graphics resources

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->editRenderer()->clearCache();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();

	if ( _displayOctree )
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	_pipeline->editRenderer()->dynamicUpdateState() = !_pipeline->editRenderer()->dynamicUpdateState();
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	_pipeline->editRenderer()->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	float3 lightPos = make_float3( pX, pY, pZ );
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &lightPos, sizeof( lightPos ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Update the associated transfer function
 *
 * @param pData the new transfer function data
 * @param pSize the size of the transfer function
 ******************************************************************************/
void SampleCore::updateTransferFunction( float* pData, unsigned int pSize )
{
	assert( _transferFunction != NULL );
	if ( _transferFunction != NULL )
	{
		// Apply modifications on transfer function's internal data
		float4* tf = _transferFunction->editData();
		unsigned int size = _transferFunction->getResolution();
		assert( size == pSize );
		for ( unsigned int i = 0; i < size; ++i )
		{
			tf[ i ] = make_float4( pData[ 4 * i ], pData[ 4 * i + 1 ], pData[ 4 * i + 2 ], pData[ 4 * i + 3 ] );
		}

		// Apply modifications on device memory
		_transferFunction->updateDeviceMemory();

		// Update cache because transfer function is applied during Producer stage
		// and not in real-time in during Sheder stage.
		_pipeline->editRenderer()->clearCache();
	}
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::initializePipeline()
{
	//// Here we compute the size of node and brick pools.
	//size_t voxelFullSize = GvCore::DataTotalChannelSize< DataType >::value;

	//size_t nodePoolNumElems = NODEPOOL_MEMSIZE / sizeof( GvStructure::OctreeNode );
	//size_t brickPoolNumElems = BRICKPOOL_MEMSIZE / voxelFullSize;

	//float nodePoolResF = powf( static_cast< float >( nodePoolNumElems ), 1.0f / 3.0f );
	//uint nodePoolResAxis = static_cast< uint >( ceil( nodePoolResF ) );
	//uint3 nodePoolRes = make_uint3( nodePoolResAxis );

	//float brickPoolResF = powf( static_cast< float >( brickPoolNumElems ), 1.0f / 3.0f );
	//uint brickPoolResAxis = static_cast< uint >( /*ceil*/( brickPoolResF ) );
	//uint3 brickPoolRes = make_uint3( brickPoolResAxis );

	//std::cout << "\nvoxelSize " << voxelFullSize << std::endl;
	//std::cout << "nodePoolRes: " << nodePoolRes << std::endl;
	//std::cout << "brickPoolRes: " << brickPoolRes << std::endl;

	// Pipeline creation
	_pipeline = new PipelineType();
	ProducerType* producer = new ProducerType();
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, producer, shader );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );

	// Graphics environment creation
	_graphicsEnvironment = new GvCommonGraphicsPass();

	return true;
}

/******************************************************************************
 * Finalize the GigaVoxels pipeline
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::finalizePipeline()
{
	// Free memory
	delete _pipeline;
	_pipeline = NULL;

	delete _graphicsEnvironment;
	_graphicsEnvironment = NULL;
	
	return true;
}

/******************************************************************************
 * Initialize the 3D model
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::initialize3DModel()
{
	if ( false )
	{
		// Upload the texture.
		// It contains a 4 components 3D float data :
		// - normal [ 3 components ]
		// - distance (signed distance field) [ 1 component ]
		QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
		QString filename = dataRepository + QDir::separator() + QString( "Voxels" ) + QDir::separator() + QString( "vd4" ) + QDir::separator() + QString( "bunny.vdCube3D4" );
		QFileInfo fileInfo( filename );
		if ( ( ! fileInfo.isFile() ) || ( ! fileInfo.isReadable() ) )
		{
			// Idea
			// Maybe use Qt function : bool QFileInfo::permission ( QFile::Permissions permissions ) const

			// TO DO
			// Handle error : free memory and exit
			// ...
			std::cout << "ERROR. Check filename : " << filename.toLatin1().constData() << std::endl;
		}

		// Create 3D model
		_signedDistanceField = new VolumeData::vdCube3D4( filename.toStdString() );
		assert( _signedDistanceField != NULL );
		if ( _signedDistanceField == NULL )
		{
			// TO DO
			// Handle error
			// ...

			return false;
		}

		// Initialize 3D model
		_signedDistanceField->initialize();
	
		// Bind the 3D model's internal data to the texture reference that will be used on device code,
		// and set texture parameters :
		// ---- access with normalized texture coordinates
		// ---- linear interpolation
		// ---- wrap texture coordinates
		_signedDistanceField->bindToTextureReference( &volumeTex, "volumeTex", true, hipFilterModeLinear, hipAddressModeWrap );
	}
    else
	{
			// Upload the texture.
		// It contains a 4 components 3D float data :
		// - normal [ 3 components ]
		// - distance (signed distance field) [ 1 component ]
		QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
		QString filename = dataRepository + QDir::separator() + QString( "Voxels" ) + QDir::separator() + QString( "vd4" ) + QDir::separator() + QString( "bunny.sdf" );
		QFileInfo fileInfo( filename );
		if ( ( ! fileInfo.isFile() ) || ( ! fileInfo.isReadable() ) )
		{
			// Idea
			// Maybe use Qt function : bool QFileInfo::permission ( QFile::Permissions permissions ) const

			// TO DO
			// Handle error : free memory and exit
			// ...
			std::cout << "ERROR. Check filename : " << filename.toLatin1().constData() << std::endl;
		}

		// Create 3D model
		VolumeData::sdfReader* sdfReaderTest = new VolumeData::sdfReader( filename.toStdString() );
		assert( sdfReaderTest != NULL );
		if ( sdfReaderTest == NULL )
		{
			// TO DO
			// Handle error
			// ...

			return false;
		}

		// Initialize 3D model
		sdfReaderTest->initialize();
	
		// Bind the 3D model's internal data to the texture reference that will be used on device code,
		// and set texture parameters :
		// ---- access with normalized texture coordinates
		// ---- linear interpolation
		// ---- wrap texture coordinates
		sdfReaderTest->bindToTextureReference( &volumeTex, "volumeTex", true, hipFilterModeLinear, hipAddressModeWrap );
	}

	return true;
}

/******************************************************************************
 * Finalize the 3D model
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::finalize3DModel()
{
	delete _signedDistanceField;

	return true;
}

/******************************************************************************
 * Initialize the transfer function
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::initializeTransferFunction()
{
	// Create the transfer function
	_transferFunction = new GvUtils::GvTransferFunction();
	assert( _transferFunction != NULL );
	if ( _transferFunction == NULL )
	{
		// TO DO
		// Handle error
		// ...

		return false;
	}

	// Initialize transfer fcuntion with a resolution of 256 elements
	_transferFunction->create( 256 );

	// Bind the transfer function's internal data to the texture reference that will be used on device code
	_transferFunction->bindToTextureReference( &transferFunctionTexture, "transferFunctionTexture", true, hipFilterModeLinear, hipAddressModeClamp );
	
	return true;
}

/******************************************************************************
 * Finalize the transfer function
 *
 * @return flag to tell wheter or not it succeeded
 ******************************************************************************/
bool SampleCore::finalizeTransferFunction()
{
	// Free memory
	delete _transferFunction;

	return true;
}

/******************************************************************************
 * Finalize graphics resources
 *
 * @return flag to tell wheter or not it succeeded
******************************************************************************/
bool SampleCore::finalizeGraphicsResources()
{
	if ( _depthBuffer )
	{
		glDeleteBuffers( 1, &_depthBuffer );
	}

	if ( _colorTex )
	{
		glDeleteTextures( 1, &_colorTex );
	}
	if ( _depthTex )
	{
		glDeleteTextures( 1, &_depthTex );
	}

	if ( _frameBuffer )
	{
		glDeleteFramebuffers( 1, &_frameBuffer );
	}

	return true;
}
